#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <hip/hip_runtime.h>

__global__ void addKernel(int* a, int* b, int* c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

void add_cuda(pybind11::array_t<int> a, pybind11::array_t<int> b, pybind11::array_t<int> c) {
    auto buf_a = a.request(), buf_b = b.request(), buf_c = c.request();
    int* d_a, *d_b, *d_c;

    int size = buf_a.shape[0];
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    hipMemcpy(d_a, buf_a.ptr, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, buf_b.ptr, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, size>>>(d_a, d_b, d_c);

    hipMemcpy(buf_c.ptr, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

PYBIND11_MODULE(example, m) {
    m.def("add_cuda", &add_cuda, "A function that adds two arrays using CUDA");
}
