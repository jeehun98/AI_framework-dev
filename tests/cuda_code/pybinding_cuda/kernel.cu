#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

// CUDA 커널 함수
__global__ void vector_add(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA 커널을 호출하는 함수
void add_vectors(float* a, float* b, float* c, int n) {
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    vector_add<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

// Pybind11 래퍼 함수
PYBIND11_MODULE(kernel, m) {
    m.def("add_vectors", &add_vectors, "Add two vectors using CUDA");
}
