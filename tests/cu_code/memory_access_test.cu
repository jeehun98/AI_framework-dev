#include <iostream>
#include <hip/hip_runtime.h>

#define N (1024 * 1024 * 10)  // 10M 요소
#define THREADS_PER_BLOCK 256
#define REPEATS 1000

__global__ void coalesced_read(float* input, float* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        output[idx] = input[idx];
}

__global__ void non_coalesced_read(float* input, float* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        output[idx] = input[(idx * 128) % N];  // 큰 stride로 non-coalesced 접근
}

template<typename Kernel>
void benchmark(const char* label, Kernel kernel, float* input, float* output) {
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    hipEventRecord(start);
    for (int i = 0; i < REPEATS; ++i) {
        kernel<<<blocks, THREADS_PER_BLOCK>>>(input, output);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float elapsed;
    hipEventElapsedTime(&elapsed, start, end);

    // 최적화 방지를 위한 출력 체크
    float* host_output = new float[10];
    hipMemcpy(host_output, output, sizeof(float) * 10, hipMemcpyDeviceToHost);
    float checksum = 0;
    for (int i = 0; i < 10; ++i)
        checksum += host_output[i];
    delete[] host_output;

    std::cout << label << " Time over " << REPEATS << " runs: " << elapsed << " ms, Checksum: " << checksum << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(end);
}

int main() {
    float *input, *output;
    hipMalloc(&input, N * sizeof(float));
    hipMalloc(&output, N * sizeof(float));

    // 간단한 초기화 (모든 값 1.0f)
    float* host_input = new float[N];
    for (int i = 0; i < N; ++i) host_input[i] = 1.0f;
    hipMemcpy(input, host_input, N * sizeof(float), hipMemcpyHostToDevice);
    delete[] host_input;

    benchmark("Coalesced", coalesced_read, input, output);
    benchmark("Non-Coalesced", non_coalesced_read, input, output);

    hipFree(input);
    hipFree(output);
    return 0;
}
