#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024 * 1024 // 데이터 크기

__global__ void kernel(float *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        data[idx] += 1.0f; // 간단한 작업
    }
}

int main() {
    float *d_data;
    hipMalloc(&d_data, N * sizeof(float));

    int blockSizes[] = {32, 64, 128, 256, 512, 1024};
    for (int blockSize : blockSizes) {
        int gridSize = (N + blockSize - 1) / blockSize;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        kernel<<<gridSize, blockSize>>>(d_data);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Block Size: %d, Time: %.3f ms\n", blockSize, milliseconds);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    hipFree(d_data);
    return 0;
}
