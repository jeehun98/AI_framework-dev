#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

__global__ void matmul_backward_input_shared(const float* __restrict__ d_out,
                                             const float* __restrict__ W_T,
                                             float* __restrict__ d_input,
                                             int M, int N, int K) {
    __shared__ float d_out_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float W_T_tile[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float sum = 0.0f;

    for (int ph = 0; ph < (N + TILE_WIDTH - 1) / TILE_WIDTH; ++ph) {
        int tiled_col = ph * TILE_WIDTH + threadIdx.x;
        int tiled_row = ph * TILE_WIDTH + threadIdx.y;

        d_out_tile[threadIdx.y][threadIdx.x] = (row < M && tiled_col < N)
            ? d_out[row * N + tiled_col] : 0.0f;

        W_T_tile[threadIdx.x][threadIdx.y] = (col < K && tiled_row < N)
            ? W_T[tiled_row * K + col] : 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            sum += d_out_tile[threadIdx.y][k] * W_T_tile[k][threadIdx.x];

        __syncthreads();
    }

    if (row < M && col < K) {
        d_input[row * K + col] = sum;

        if (row == 0 && col == 0) {
            // printf("[matmul_backward_input] d_input[0] = %f\n", sum);
        }
    }
}

__global__ void matmul_backward_weight_shared(const float* __restrict__ input_T,  // [K x M]
                                              const float* __restrict__ d_out,    // [M x N]
                                              float* __restrict__ d_weight,       // [K x N]
                                              int K, int N, int M) {
    __shared__ float input_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float d_out_tile[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;  // K
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;  // N

    float sum = 0.0f;

    for (int ph = 0; ph < (M + TILE_WIDTH - 1) / TILE_WIDTH; ++ph) {
        int tiled_col = ph * TILE_WIDTH + threadIdx.x;

        input_tile[threadIdx.y][threadIdx.x] = (row < K && tiled_col < M)
            ? input_T[row * M + tiled_col] : 0.0f;

        d_out_tile[threadIdx.y][threadIdx.x] = (tiled_col < M && col < N)
            ? d_out[tiled_col * N + col] : 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            sum += input_tile[threadIdx.y][k] * d_out_tile[k][threadIdx.x];

        __syncthreads();
    }

    if (row < K && col < N) {
        d_weight[row * N + col] = sum;

        if (row == 0 && col == 0) {
            // printf("[matmul_bw_weight] d_weight[0] = %f, input_T[0] = %f, d_out[0] = %f\n", sum, input_T[0], d_out[0]);
        }
    }
}

__global__ void add_backward_bias(const float* d_out, float* d_bias, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < cols) {
        float sum = 0.0f;
        for (int i = 0; i < rows; ++i)
            sum += d_out[i * cols + col];
        d_bias[col] = sum;

        if (col == 0) {
            // printf("[add_backward_bias] d_bias[0] = %f\n", sum);
        }
    }
}

__global__ void add_backward_input(const float* d_out, float* d_input, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        d_input[i] = d_out[i];

        if (i == 0) {
            // printf("[add_backward_input] d_input[0] = %f (from d_out[0] = %f)\n", d_input[i], d_out[i]);
        }
    }
}

__global__ void fill_gradient(float* grad, int total_size, float value) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_size) {
        grad[i] = value;
        if (i == 0) {
            // printf("[fill_gradient] grad[0] = %f\n", value);
        }
    }
}

__global__ void matmul_backward_input_simple(const float* __restrict__ d_out,
                                             const float* __restrict__ W_T,
                                             float* __restrict__ d_input,
                                             int M, int N, int K) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= M * K) return;

    int row = idx / K;
    int col = idx % K;

    float sum = 0.0f;
    for (int n = 0; n < N; ++n) {
        sum += d_out[row * N + n] * W_T[n * K + col];
    }

    // ✅ atomic 제거 (단일 쓰레드만 해당 위치 접근하므로 안전)
    d_input[row * K + col] = sum;

    if (idx == 0) {
        // printf("[matmul_bw_input_simple] M=%d, N=%d, K=%d | d_out[0]=%.6f, W_T[0]=%.6f, sum=%.6f\n", M, N, K, d_out[0], W_T[0], sum);
    }
}
