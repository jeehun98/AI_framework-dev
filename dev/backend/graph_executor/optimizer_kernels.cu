#include "hip/hip_runtime.h"
// optimizer_kernels.cu
#include "optimizer_types.cuh"
#include <hip/hip_runtime.h>
#include <math.h>          // isfinite, sqrtf, powf, fminf/fmaxf
#include "logging_config.h"

#include "optimizer_config.cuh"
#include "optimizer_kernels.cuh"


// ===== 옵션 =====
#ifndef GRAD_CLIP_ENABLE          // 값 클리핑(절댓값 기준)
#define GRAD_CLIP_ENABLE 0
#endif
#ifndef GRAD_CLIP_THRESH
#define GRAD_CLIP_THRESH 1e4f
#endif

#ifndef GLOBAL_NORM_CLIP_ENABLE   // 글로벌 L2 노름 클리핑
#define GLOBAL_NORM_CLIP_ENABLE 0
#endif

#ifndef WEIGHT_DECAY_ENABLE       // Decoupled WD(AdamW/SGD-WD)
#define WEIGHT_DECAY_ENABLE 0
#endif

#ifndef NESTEROV_ENABLE           // 모멘텀의 Nesterov 모드
#define NESTEROV_ENABLE 0
#endif

#ifndef AMSGRAD_ENABLE            // Adam에서 vhat의 최대치 유지
#define AMSGRAD_ENABLE 0
#endif

#ifndef DEBUG_KERNEL
#define DEBUG_KERNEL 0
#endif

// ===== 공용 유틸 =====
__device__ __forceinline__ float value_clip(float g) {
#if GRAD_CLIP_ENABLE
    // 값 클리핑: 지나치게 큰 gradient를 [-T, +T]로 clamp
    const float T = GRAD_CLIP_THRESH;
    if (!isfinite(g)) return 0.0f;
    if (g >  T) return  T;
    if (g < -T) return -T;
#endif
    return g;
}

// ===== 글로벌 노름 클리핑(2-pass)용 리덕션 =====
#if GLOBAL_NORM_CLIP_ENABLE
__global__ void grad_sqsum_kernel(const float* __restrict__ grad, double* __restrict__ partial, int n) {
    extern __shared__ double s[];
    double sum = 0.0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float g = grad[i];
        if (isfinite(g)) {
            // 값 클리핑은 스케일 전에 하도록 동일 함수 사용
            g = value_clip(g);
            sum += (double)g * (double)g;
        }
    }
    s[threadIdx.x] = sum;
    __syncthreads();

    // block reduce
    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) s[threadIdx.x] += s[threadIdx.x + stride];
        __syncthreads();
    }
    if (threadIdx.x == 0) partial[blockIdx.x] = s[0];
}

__global__ void scale_grad_kernel(const float* __restrict__ grad_in, float* __restrict__ grad_out, float scale, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float g = value_clip(grad_in[i]);
        grad_out[i] = g * scale;
    }
}
#endif // GLOBAL_NORM_CLIP_ENABLE

// ===== SGD =====
__global__ void sgd_kernel(float* __restrict__ param,
                           const float* __restrict__ grad,
#if WEIGHT_DECAY_ENABLE
                           float weight_decay,
#endif
                           float lr, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float g = value_clip(grad[i]);
        if (!isfinite(g)) continue;

#if WEIGHT_DECAY_ENABLE
        // Decoupled WD: p = p - lr*(g) - lr*wd*p
        float p = param[i];
        p -= lr * g;
        p -= lr * weight_decay * p;
        param[i] = p;
#else
        param[i] -= lr * g;
#endif

#if DEBUG_KERNEL
        if (i == 0) KPRINTF("[SGD] lr=%g, g0=%g, p0=%g\n", lr, g, param[i]);
#endif
    }
}

// ===== Momentum (classic/Nesterov) =====
__global__ void momentum_kernel(float* __restrict__ param,
                                const float* __restrict__ grad,
                                float* __restrict__ velocity,
#if WEIGHT_DECAY_ENABLE
                                float weight_decay,
#endif
                                float lr, float beta, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float g = value_clip(grad[i]);
        if (!isfinite(g)) continue;

#if WEIGHT_DECAY_ENABLE
        // decoupled WD
        float p = param[i];
        p -= lr * weight_decay * p;
        param[i] = p;
#endif

        float v = beta * velocity[i] + g;
        velocity[i] = v;

#if NESTEROV_ENABLE
        float upd = lr * (beta * v + g);  // Nesterov
#else
        float upd = lr * v;               // classic
#endif
        if (!isfinite(upd)) continue;
        param[i] -= upd;

#if DEBUG_KERNEL
        if (i == 0) KPRINTF("[MOMENTUM] lr=%g beta=%g | g0=%g v0=%g p0=%g\n", lr, beta, g, v, param[i]);
#endif
    }
}

// ===== Adam / AdamW (+ AMSGrad) =====
__global__ void adam_kernel(float* __restrict__ param,
                            const float* __restrict__ grad,
                            float* __restrict__ m,
                            float* __restrict__ v,
#if AMSGRAD_ENABLE
                            float* __restrict__ vhat_max,
#endif
#if WEIGHT_DECAY_ENABLE
                            float weight_decay,
#endif
                            float lr, float beta1, float beta2, float eps,
                            int t, int n) {
    const int t_eff = (t < 1) ? 1 : t;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float g = value_clip(grad[i]);
        if (!isfinite(g)) continue;

        float mi = beta1 * m[i] + (1.0f - beta1) * g;
        float vi = beta2 * v[i] + (1.0f - beta2) * g * g;
        m[i] = mi;
        v[i] = vi;

        // bias correction
        float m_hat = mi / fmaxf(1.0f - powf(beta1, (float)t_eff), 1e-12f);
        float v_hat = vi / fmaxf(1.0f - powf(beta2, (float)t_eff), 1e-12f);

#if AMSGRAD_ENABLE
        float vmax = fmaxf(v_hat, vhat_max[i]);
        vhat_max[i] = vmax;
        v_hat = vmax;
#endif
        float denom = sqrtf(fmaxf(v_hat, 1e-12f)) + eps;
        float step  = lr * (m_hat / denom);
        if (!isfinite(step)) continue;

#if WEIGHT_DECAY_ENABLE
        // AdamW: decoupled WD
        float p = param[i];
        p -= lr * weight_decay * p;
        p -= step;
        param[i] = p;
#else
        param[i] -= step;
#endif

#if DEBUG_KERNEL
        if (i == 0 && (t_eff % 100 == 0)) {
            KPRINTF("[ADAM] t=%d lr=%g b1=%g b2=%g eps=%g | g0=%g m=%g v=%g m^=%g v^=%g step=%g p0=%g\n",
                t_eff, lr, beta1, beta2, eps, g, mi, vi, m_hat, v_hat, step, param[i]);
        }
#endif
    }
}

// ===== Host Launcher =====
void optimizer_update_cuda(
    float* param,
    const float* grad,     // ★ const 유지
    float* velocity,
    float* m,
    float* v,
#if AMSGRAD_ENABLE
    float* vhat_max,
#endif
    float lr, float beta1, float beta2, float eps,
#if WEIGHT_DECAY_ENABLE
    float weight_decay,
#endif
    int size,
    OptimizerType opt_type,
    int timestep,
    hipStream_t stream     // ★ 정의에도 stream 추가
){
    const int threads = 256;
    const int blocks  = (size + threads - 1) / threads;

#if GLOBAL_NORM_CLIP_ENABLE
    // 1) grad L2 norm 계산 → 2) scale 적용된 임시 grad_buf 생성
    //    (메모리 여유 없으면 in-place 스케일링 커널로 바꿔도 됨)
    static float* grad_scaled = nullptr;
    static int    grad_cap = 0;
    if (grad_cap < size) {
        if (grad_scaled) hipFree(grad_scaled);
        hipMalloc(&grad_scaled, size * sizeof(float));
        grad_cap = size;
    }

    // partial sums
    int redBlocks = min(blocks, 1024);
    double* d_partial = nullptr;
    hipMalloc(&d_partial, redBlocks * sizeof(double));

    size_t shmem = threads * sizeof(double);
    grad_sqsum_kernel<<<redBlocks, threads, shmem>>>(grad, d_partial, size);

    // host reduce
    double* h_partial = (double*)malloc(redBlocks * sizeof(double));
    hipMemcpy(h_partial, d_partial, redBlocks * sizeof(double), hipMemcpyDeviceToHost);
    double sum = 0.0;
    for (int i = 0; i < redBlocks; ++i) sum += h_partial[i];
    free(h_partial);
    hipFree(d_partial);

    double norm = sqrt(sum + 1e-30);
    float scale = 1.0f;
    // 일반적으로 clip_threshold는 학습 코드 상위에서 전달
    const float clip_threshold = GRAD_CLIP_THRESH; // 재사용
    if (norm > (double)clip_threshold) {
        scale = (float)((double)clip_threshold / norm);
    }
    scale_grad_kernel<<<blocks, threads>>>(grad, grad_scaled, scale, size);
    const float* gptr = grad_scaled;
#else
    const float* gptr = grad;
#endif

    switch (opt_type) {
        case OptimizerType::SGD:
            sgd_kernel<<<blocks, threads>>>(param, gptr,
#if WEIGHT_DECAY_ENABLE
                weight_decay,
#endif
                lr, size);
            break;

        case OptimizerType::MOMENTUM:
            if (!velocity) return;
            momentum_kernel<<<blocks, threads>>>(param, gptr, velocity,
#if WEIGHT_DECAY_ENABLE
                weight_decay,
#endif
                lr, beta1 /*as beta*/, size);
            break;

        case OptimizerType::ADAM:
            if (!m || !v) return;
            adam_kernel<<<blocks, threads>>>(param, gptr, m, v,
#if AMSGRAD_ENABLE
                vhat_max,
#endif
#if WEIGHT_DECAY_ENABLE
                weight_decay,
#endif
                lr, beta1, beta2, eps, timestep, size);
            break;

        default:
            return;
    }

#if DEBUG_KERNEL
    hipDeviceSynchronize();
#endif
}
