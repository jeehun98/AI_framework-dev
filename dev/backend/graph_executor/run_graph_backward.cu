#include "hip/hip_runtime.h"
// run_graph_backward.cu (updated)
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <unordered_map>
#include <hipblas.h>                    // ✅ cuBLAS

#include "run_graph.cuh"
// ↓ 아래 3개는 더이상 필요 없음: backward matmul/transpose를 cuBLAS로 대체
// #include "backward_kernels_optimized.cuh"
// #include "transpose.cuh"
#include "activation_ops.cuh"
#include "cnn_kernels.cuh"
#include "op_structs.cuh"
#include "loss_kernels.cuh"

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

__global__ void add_inplace(float* dst, const float* src, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) dst[i] += src[i];
}

static inline void checkCudaLast(const char* where) {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        std::fprintf(stderr, "[CUDA][ERR] %s: %s\n", where, hipGetErrorString(e));
    }
}
static inline void checkCudaSync(const char* where) {
    hipError_t e = hipDeviceSynchronize();
    if (e != hipSuccess) {
        std::fprintf(stderr, "[CUDA][SYNC] %s: %s\n", where, hipGetErrorString(e));
    }
}

// ✅ cuBLAS 에러 헬퍼
static inline void CUBLAS_CHECK(hipblasStatus_t s, const char* where) {
    if (s != HIPBLAS_STATUS_SUCCESS) {
        std::fprintf(stderr, "[cuBLAS][ERR] %s: code=%d\n", where, int(s));
    }
}

/**
 * Row-major 행렬을 대상으로 하는 얇은 GEMM 래퍼.
 * 우리가 원하는 C(MxN)=A(MxK)*B(KxN)을 column-major인 cuBLAS로 호출하기 위해
 * 'B, A' 순서로 뒤집어 넣는다.
 */
static inline void gemm_rm(hipblasHandle_t h,
                           bool transA, bool transB,
                           int M, int N, int K,
                           const float* A, int lda,   // row-major: lda=열 개수
                           const float* B, int ldb,
                           float* C, int ldc,
                           float alpha=1.0f, float beta=0.0f)
{
    // row-major에서의 전치 플래그
    hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    // column-major로 바꿔 호출: C^T = B^T * A^T
    // => m=N, n=M, k=K, (opB,opA), (B,A), leading dims는 row-major의 열 개수
    CUBLAS_CHECK(
        hipblasSgemm(h,
                    opB, opA,
                    /*m=*/N, /*n=*/M, /*k=*/K,
                    &alpha,
                    B, ldb,
                    A, lda,
                    &beta,
                    C, ldc),
        "hipblasSgemm");
}

// ==== ADD backward: dX = dY, dB = sum_rows(dY) ===============================
static __global__ void add_backward_input(const float* __restrict__ grad_out,
                                          float* __restrict__ grad_in,
                                          int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) grad_in[i] = grad_out[i];  // dX = dY
}

// grad_bias[col] = sum_{row=0..rows-1} grad_out[row, col]
static __global__ void add_backward_bias(const float* __restrict__ grad_out,
                                         float* __restrict__ grad_bias,
                                         int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= cols) return;

    float s = 0.f;
    // grad_out 은 row-major [rows, cols]
    for (int r = 0; r < rows; ++r) {
        s += grad_out[r * cols + col];
    }
    grad_bias[col] = s;  // 호출 측에서 batch는 바깥 루프로 누적(add_inplace) 처리
}


void run_graph_backward(
    const std::vector<OpStruct>& E,
    std::unordered_map<std::string, float*>& tensors,
    std::unordered_map<std::string, Shape>& shapes,
    std::unordered_map<std::string, float*>& gradients,
    const std::string& final_output_id,  // ← 반드시 Activation output의 ID여야 함
    int batch_size)
{
    // cuBLAS 핸들
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle), "hipblasCreate");

    std::string grad_start_id = final_output_id;

    // 1) LOSS 처리: dL/dy_pred 생성
    if (!E.empty() && E.back().op_type == LOSS) {
        const OpStruct& loss_op = E.back();

        std::string loss_type = loss_op.extra_params.loss_type;
        std::string label_id  = loss_op.extra_params.label_id;
        const float* y_true   = tensors[label_id];
        const float* y_pred   = tensors[loss_op.input_id];  // ★ Activation 출력이어야 함

        Shape shp = shapes[loss_op.input_id];  // per-sample
        int per_sample = shp.rows * shp.cols;
        int sz = batch_size * per_sample;

        float* dL_dy = nullptr;
        hipMalloc(&dL_dy, sz * sizeof(float));

        if (loss_type == "bce") {
            bce_loss_backward<<<(sz + 255)/256, 256>>>(y_true, y_pred, dL_dy, sz, batch_size);
            checkCudaLast("bce_loss_backward");
            checkCudaSync("bce_loss_backward");
        } else if (loss_type == "mse") {
            mse_loss_backward<<<(sz + 255)/256, 256>>>(y_true, y_pred, dL_dy, sz);
            checkCudaLast("mse_loss_backward");
            checkCudaSync("mse_loss_backward");
        } else {
            std::fprintf(stderr, "[LOSS][BW] unsupported: %s\n", loss_type.c_str());
        }

        grad_start_id = loss_op.input_id;
        gradients[loss_op.input_id] = dL_dy;
    }

    // 2) 나머지 역전파
    for (auto it = E.rbegin(); it != E.rend(); ++it) {
        const OpStruct& op = *it;
        if (op.op_type == LOSS) continue;

        float* input = tensors[op.input_id];
        float* param = (!op.param_id.empty() && tensors.count(op.param_id))
                         ? tensors[op.param_id] : nullptr;
        float* grad_out_full = gradients[op.output_id];
        if (!grad_out_full && op.op_type != FLATTEN) continue;

        Shape in_shape  = shapes[op.input_id];
        Shape out_shape = shapes[op.output_id];
        if (out_shape.rows == 0 || out_shape.cols == 0) out_shape = in_shape;

        const int in_rows  = in_shape.rows,  in_cols  = in_shape.cols;   // X: [M,K]
        const int out_rows = out_shape.rows, out_cols = out_shape.cols;  // Y: [M,N]
        const int in_size  = in_rows * in_cols;
        const int out_size = out_rows * out_cols;

        const size_t in_stride  = (size_t)in_size;
        const size_t out_stride = (size_t)out_size;

        float* grad_input_full = nullptr;
        if (op.op_type != FLATTEN) {
            hipMalloc(&grad_input_full, (size_t)batch_size * in_size * sizeof(float));
        }

        switch (op.op_type) {
        case MATMUL: {
            if (!param) break;

            // Shapes (row-major):
            // Forward:  Y[M,N] = X[M,K] * W[K,N]
            // Backward:
            //   dX[M,K] = dY[M,N] * W^T[N,K]
            //   dW[K,N] = X^T[K,M] * dY[M,N]   (배치 전체 합)
            const int M = in_rows;
            const int K = in_cols;
            const int N = out_cols;

            // dW 누적 버퍼
            float* grad_weight = nullptr;
            hipMalloc(&grad_weight, (size_t)K * N * sizeof(float));
            hipMemset(grad_weight, 0, (size_t)K * N * sizeof(float));

            // 배치 루프
            for (int b = 0; b < batch_size; ++b) {
                float* grad_out_b   = grad_out_full   + b * out_stride; // [M,N]
                float* grad_input_b = grad_input_full + b * in_stride;  // [M,K]
                float* input_b      = input           + b * in_stride;  // [M,K]

                // dX = dY * W^T
                // lda/ldb/ldc는 row-major의 '열 개수'
                gemm_rm(handle,
                        /*transA=*/false, /*transB=*/true,
                        /*M=*/M, /*N=*/K, /*K=*/N,
                        /*A=*/grad_out_b, /*lda=*/N,
                        /*B=*/param,      /*ldb=*/N,   // W[K,N], transB=True로 W^T 사용
                        /*C=*/grad_input_b,/*ldc=*/K,
                        /*alpha=*/1.f, /*beta=*/0.f);

                // dW += X^T * dY
                gemm_rm(handle,
                        /*transA=*/true,  /*transB=*/false,
                        /*M=*/K, /*N=*/N, /*K=*/M,
                        /*A=*/input_b,    /*lda=*/K,   // X[M,K], transA=True로 X^T
                        /*B=*/grad_out_b, /*ldb=*/N,   // dY[M,N]
                        /*C=*/grad_weight,/*ldc=*/N,
                        /*alpha=*/1.f, /*beta=*/1.f);  // 누적
            }

            gradients[op.param_id] = grad_weight;  // dW
            break;
        }

        case ADD: {
            // y = x + b(row-wise), backward:
            //   dX = dY
            //   dB = sum_rows(dY), 배치 전체 합
            float* grad_bias = nullptr;
            hipMalloc(&grad_bias, (size_t)out_cols * sizeof(float));
            hipMemset(grad_bias, 0, (size_t)out_cols * sizeof(float));

            // 임시 버퍼 1개만 재사용
            float* grad_bias_b = nullptr;
            hipMalloc(&grad_bias_b, (size_t)out_cols * sizeof(float));

            for (int b = 0; b < batch_size; ++b) {
                float* grad_out_b   = grad_out_full   + b * out_stride;
                float* grad_input_b = grad_input_full + b * in_stride;

                // dX = dY
                add_backward_input<<<(out_size + 255)/256, 256>>>(grad_out_b, grad_input_b, out_size);
                checkCudaLast("add_backward_input");

                // dB_b = sum_rows(dY)
                hipMemset(grad_bias_b, 0, (size_t)out_cols * sizeof(float));
                add_backward_bias<<<(out_cols + 255)/256, 256>>>(grad_out_b, grad_bias_b, out_rows, out_cols);
                checkCudaLast("add_backward_bias");

                // 누적
                int thr = 256, blk = (out_cols + thr - 1) / thr;
                add_inplace<<<blk, thr>>>(grad_bias, grad_bias_b, out_cols);
                checkCudaLast("add_inplace grad_bias");
            }
            hipFree(grad_bias_b);

            gradients[op.param_id] = grad_bias;
            break;
        }

        case SIGMOID:
        case RELU:
        case TANH: {
            // rows' = batch_size * out_rows, cols' = out_cols
            const int rowsB = batch_size * out_rows;
            const int colsB = out_cols;

            // grad_out_full / grad_input_full / tensors[op.output_id] 는
            // 배치가 연속 저장이므로 그대로 전달하면 OK
            launch_activation_backward(
                /*grad_out=*/grad_out_full,
                /*out=*/tensors[op.output_id],
                /*grad_in=*/grad_input_full,
                rowsB, colsB, op.op_type);
            checkCudaLast("activation_backward");
            break;
        }


        case FLATTEN: {
            // 단순 전달
            gradients[op.input_id] = grad_out_full;
            continue;
        }

        default:
            break;
        }

        if (op.op_type != FLATTEN) {
            if (!grad_input_full) {
                std::fprintf(stderr, "[BW] grad_input_full is null: op=%d\n", op.op_type);
            } else {
                gradients[op.input_id] = grad_input_full;
            }
        }
    }

    hipblasDestroy(handle);
}
