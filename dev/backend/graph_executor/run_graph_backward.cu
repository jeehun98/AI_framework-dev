#include "hip/hip_runtime.h"
// run_graph_backward.cu (final, TF32 + strided-batched + fused softmax-xent)
#include <iostream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <unordered_map>
#include <hipblas.h>

#include "run_graph.cuh"
#include "activation_ops.cuh"
#include "softmax_kernels.cuh"
#include "cnn_kernels.cuh"
#include "op_structs.cuh"
#include "loss_kernels.cuh"

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

#ifndef CUDA_CHECK
#define CUDA_CHECK(x) do { hipError_t _e=(x); if(_e!=hipSuccess){ \
  fprintf(stderr,"[CUDA] %s:%d %s\n", __FILE__, __LINE__, hipGetErrorString(_e)); } } while(0)
#endif

#ifndef CUBLAS_CHECK
#define CUBLAS_CHECK(call) do { \
    hipblasStatus_t _st = (call); \
    if (_st != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "[cuBLAS] %s:%d status=%d\n", __FILE__, __LINE__, (int)_st); \
    } \
} while(0)
#endif

// === 전역 cuBLAS 핸들 재사용 ===
static hipblasHandle_t g_cublas = nullptr;
static void ensure_cublas() {
    if (!g_cublas) {
        CUBLAS_CHECK(hipblasCreate(&g_cublas));
        // 성능 우선시 TF32 활성화하려면 주석 해제
        // CUBLAS_CHECK(hipblasSetMathMode(g_cublas, HIPBLAS_TF32_TENSOR_OP_MATH));
    }
}

// 디버그/동기 헬퍼
static inline void checkCudaLast(const char* where) {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        std::fprintf(stderr, "[CUDA][ERR] %s: %s\n", where, hipGetErrorString(e));
    }
}
static inline void checkCudaSync(const char* where) {
    hipError_t e = hipDeviceSynchronize();
    if (e != hipSuccess) {
        std::fprintf(stderr, "[CUDA][SYNC] %s: %s\n", where, hipGetErrorString(e));
    }
}

// ones 벡터 채우기
__global__ void fill_kernel(float* p, float v, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) p[i] = v;
}

// ---- GEMM 래퍼 (row-major 매핑) --------------------------------------------
// 단일 GEMM (row-major) TF32
static inline void gemm_rm_tf32(
    hipblasHandle_t h,
    bool transA, bool transB,
    int M, int N, int K,
    const float* A, int lda,
    const float* B, int ldb,
    float* C, int ldc,
    float alpha=1.f, float beta=0.f)
{
    hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(
        hipblasGemmEx(
            h,
            /*opB,opA*/ opB, opA,
            /*m,n,k*/   N,   M,   K,
            &alpha,
            /*B*/ B, HIP_R_32F, ldb,
            /*A*/ A, HIP_R_32F, lda,
            &beta,
            /*C*/ C, HIP_R_32F, ldc,
            /*computeType*/ HIPBLAS_COMPUTE_32F_FAST_TF32,
            /*algo*/ CUBLAS_GEMM_DEFAULT_TENSOR_OP
        )
    );
}

// StridedBatched GEMM (row-major) TF32
static inline void gemm_rm_strided_batched_tf32(
    hipblasHandle_t h,
    bool transA, bool transB,
    int M, int N, int K,
    const float* A, int lda, long long strideA,
    const float* B, int ldb, long long strideB,
    float* C, int ldc, long long strideC,
    int batch,
    float alpha=1.f, float beta=0.f)
{
    hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(
        hipblasGemmStridedBatchedEx(
            h,
            /*opB,opA*/ opB, opA,
            /*m,n,k*/   N,   M,   K,
            &alpha,
            /*B*/ B, HIP_R_32F, ldb, strideB,
            /*A*/ A, HIP_R_32F, lda, strideA,
            &beta,
            /*C*/ C, HIP_R_32F, ldc, strideC,
            /*batch*/ batch,
            /*computeType*/ HIPBLAS_COMPUTE_32F_FAST_TF32,
            /*algo*/ CUBLAS_GEMM_DEFAULT_TENSOR_OP
        )
    );
}
// -----------------------------------------------------------------------------

// 활성화 매핑
static inline int map_act_type(int op_type) {
    switch (op_type) {
        case SIGMOID:    return ACT_SIGMOID;
        case RELU:       return ACT_RELU;
        case TANH:       return ACT_TANH;
        case LEAKY_RELU: return ACT_LEAKY;
        case ELU:        return ACT_ELU;
        case GELU:       return ACT_GELU;
        case SILU:       return ACT_SILU;
        default:         return ACT_IDENTITY;
    }
}

void run_graph_backward(
    const std::vector<OpStruct>& E,
    std::unordered_map<std::string, float*>& tensors,
    std::unordered_map<std::string, Shape>& shapes,
    std::unordered_map<std::string, float*>& gradients,
    const std::string& final_output_id,  // ← 보통 activation output의 ID
    int batch_size)
{
    ensure_cublas();

    std::string grad_start_id = final_output_id;
    bool fused_softmax = false;
    std::string fused_softmax_in_id, fused_softmax_out_id;

    // 1) LOSS backward: dL/dy_pred (혹은 fused면 dL/dz) 생성
    if (!E.empty() && E.back().op_type == LOSS) {
        const OpStruct& loss_op = E.back();
        const std::string loss_type = loss_op.extra_params.loss_type;
        const std::string label_id  = loss_op.extra_params.label_id;

        const float* y_true = tensors[label_id];
        const float* y_pred = tensors[loss_op.input_id];

        Shape shp = shapes[loss_op.input_id];
        const int C = shp.cols;
        const int rows_per_sample = shp.rows;          // 보통 1
        const int B = batch_size * rows_per_sample;
        const int N = B * C;

        hipStream_t stream = 0;

        // 직전 op이 SOFTMAX인지 확인 (fused 조건)
        const OpStruct* prev = nullptr;
        if (E.size() >= 2) {
            const OpStruct& cand = E[E.size()-2];
            if (cand.op_type == SOFTMAX && cand.output_id == loss_op.input_id) {
                prev = &cand;
            }
        }

        if (loss_type == "cce" && prev) {
            // ✅ fused: ∂L/∂z = (p - y) / B, SOFTMAX는 스킵
            float* dL_dz = nullptr;
            CUDA_CHECK(hipMalloc(&dL_dz, (size_t)N * sizeof(float)));
            launch_softmax_xent_fused_backward(
                /*y_prob*/ y_pred,
                /*y_true*/ y_true,
                /*grad_z*/ dL_dz,
                /*B*/ B, /*C*/ C, stream
            );
            checkCudaLast("launch_softmax_xent_fused_backward");
            checkCudaSync("softmax_xent_fused_backward sync");

            fused_softmax = true;
            fused_softmax_in_id  = prev->input_id;   // z
            fused_softmax_out_id = prev->output_id;  // p
            grad_start_id = prev->input_id;
            gradients[prev->input_id] = dL_dz;
        } else {
            // 일반 경로: dL/dY(또는 dL/da) 생성
            float* dL_dy = nullptr;
            CUDA_CHECK(hipMalloc(&dL_dy, (size_t)N * sizeof(float)));

            if (loss_type == "bce") {
                launch_bce_loss_backward(y_true, y_pred, dL_dy, N, B, stream);
                checkCudaLast("launch_bce_loss_backward");
                checkCudaSync("bce_backward sync");
            } else if (loss_type == "mse") {
                launch_mse_loss_backward(y_true, y_pred, dL_dy, N, stream);
                checkCudaLast("launch_mse_loss_backward");
                checkCudaSync("mse_backward sync");
            } else if (loss_type == "cce") {
                // softmax 출력(확률)에 대한 dL/dY = -(y/p)/B
                launch_cce_loss_backward(y_true, y_pred, dL_dy, B, C, stream);
                checkCudaLast("launch_cce_loss_backward");
                checkCudaSync("cce_backward sync");
            } else {
                std::fprintf(stderr, "[LOSS][BW] unsupported: %s\n", loss_type.c_str());
            }

            grad_start_id = loss_op.input_id;
            gradients[loss_op.input_id] = dL_dy;
        }
    }

    // 2) 나머지 역전파
    for (auto it = E.rbegin(); it != E.rend(); ++it) {
        const OpStruct& op = *it;
        if (op.op_type == LOSS) continue;

        // ✅ fused면 해당 SOFTMAX 노드는 스킵
        if (fused_softmax && op.op_type == SOFTMAX && op.output_id == fused_softmax_out_id) {
            continue;
        }

        float* input = tensors[op.input_id];
        float* param = (!op.param_id.empty() && tensors.count(op.param_id))
                         ? tensors[op.param_id] : nullptr;
        float* grad_out_full = gradients[op.output_id];
        if (!grad_out_full && op.op_type != FLATTEN) continue;

        Shape in_shape  = shapes[op.input_id];
        Shape out_shape = shapes[op.output_id];
        if (out_shape.rows == 0 || out_shape.cols == 0) out_shape = in_shape;

        const int M = out_shape.rows;   // == in_shape.rows
        const int N = out_shape.cols;
        const int K = in_shape.cols;
        const int in_size  = in_shape.rows * in_shape.cols;
        const int out_size = out_shape.rows * out_shape.cols;

        float* grad_input_full = nullptr;
        if (op.op_type != FLATTEN) {
            CUDA_CHECK(hipMalloc(&grad_input_full, (size_t)batch_size * in_size * sizeof(float)));
        }

        switch (op.op_type) {
        case MATMUL: {
            if (!param) break; // W 없음

            // dX = dY · W^T  (B, M, K)
            gemm_rm_strided_batched_tf32(
                g_cublas,
                /*transA=*/false, /*transB=*/true,
                /*M=*/M, /*N=*/K, /*K=*/N,
                /*A =*/ grad_out_full,   /*lda =*/ N, /*strideA =*/ (long long)M * N,
                /*B =*/ param,           /*ldb =*/ N, /*strideB =*/ 0LL,
                /*C =*/ grad_input_full, /*ldc =*/ K, /*strideC =*/ (long long)M * K,
                /*batch=*/batch_size,
                /*alpha=*/1.f, /*beta=*/0.f
            );

            // dW = sum_b (X_b^T · dY_b)
            // 1) dW_tmp[b] = X_b^T(K,M) · dY_b(M,N)  →  (B, K, N)
            float* dW_tmp = nullptr;
            CUDA_CHECK(hipMalloc(&dW_tmp, (size_t)batch_size * K * N * sizeof(float)));

            gemm_rm_strided_batched_tf32(
                g_cublas,
                /*transA=*/true, /*transB=*/false,
                /*M=*/K, /*N=*/N, /*K=*/M,
                /*A =*/ input,          /*lda =*/ K, /*strideA =*/ (long long)M * K,
                /*B =*/ grad_out_full,  /*ldb =*/ N, /*strideB =*/ (long long)M * N,
                /*C =*/ dW_tmp,         /*ldc =*/ N, /*strideC =*/ (long long)K * N,
                /*batch=*/batch_size,
                /*alpha=*/1.f, /*beta=*/0.f
            );

            // 2) 배치축 합산: ones(1,B) · dW_tmp(B, K*N) → dW(1, K*N)
            float* dW_accum = nullptr;
            CUDA_CHECK(hipMalloc(&dW_accum, (size_t)K * N * sizeof(float)));

            float* onesB = nullptr;
            CUDA_CHECK(hipMalloc(&onesB, (size_t)batch_size * sizeof(float)));
            {
                int thr = 256, blk = (batch_size + thr - 1) / thr;
                fill_kernel<<<blk, thr>>>(onesB, 1.0f, batch_size);
            }

            // C(1, K*N) = A(1, B) · B(B, K*N)
            gemm_rm_tf32(
                g_cublas, false, false,
                /*M=*/1, /*N=*/(K * N), /*K=*/batch_size,
                /*A=*/onesB,     /*lda=*/batch_size,
                /*B=*/dW_tmp,    /*ldb=*/(K * N),
                /*C=*/dW_accum,  /*ldc=*/(K * N),
                1.f, 0.f
            );

            gradients[op.param_id] = dW_accum;

            CUDA_CHECK(hipFree(dW_tmp));
            CUDA_CHECK(hipFree(onesB));
            break;
        }

        case ADD: {
            // dX = dY (그대로 복사)
            const size_t bytes = (size_t)batch_size * out_size * sizeof(float);
            CUDA_CHECK(hipMemcpy(grad_input_full, grad_out_full, bytes, hipMemcpyDeviceToDevice));

            // dB = sum over batch and rows → ones(1, B*M) · dY(B*M, N)
            const int rowsB = batch_size * M;
            const int cols  = N;

            float* grad_bias = nullptr;      // [cols]
            CUDA_CHECK(hipMalloc(&grad_bias, (size_t)cols * sizeof(float)));

            float* onesR = nullptr;
            CUDA_CHECK(hipMalloc(&onesR, (size_t)rowsB * sizeof(float)));
            {
                int thr = 256, blk = (rowsB + thr - 1) / thr;
                fill_kernel<<<blk, thr>>>(onesR, 1.0f, rowsB);
            }

            gemm_rm_tf32(
                g_cublas, false, false,
                /*M=*/1, /*N=*/cols, /*K=*/rowsB,
                /*A=*/onesR,            /*lda=*/rowsB,
                /*B=*/grad_out_full,    /*ldb=*/cols,
                /*C=*/grad_bias,        /*ldc=*/cols,
                1.f, 0.f
            );

            gradients[op.param_id] = grad_bias;
            CUDA_CHECK(hipFree(onesR));
            break;
        }

        // -------- 활성화 계열: launch_activation_backward 호출 --------
        case SIGMOID:
        case RELU:
        case TANH:
        case LEAKY_RELU:
        case ELU:
        case GELU:
        case SILU:
        {
            const int rowsB = batch_size * out_shape.rows;
            const int cols  = out_shape.cols;

            const float* gout = grad_out_full;             // dL/dout
            const float* out  = tensors[op.output_id];     // f(z)
            const float* in   = tensors[op.input_id];      // z (pre-activation)
            float* gin        = grad_input_full;           // dL/din

            const int act = map_act_type(op.op_type);
            const float alpha = op.extra_params.alpha;
            const int gelu_tanh_flag = op.extra_params.gelu_tanh ? 1 : 0;

            hipStream_t stream = 0;

            launch_activation_backward(
                /*grad_out*/ gout,
                /*in      */ in,
                /*out     */ out,
                /*grad_in */ gin,
                /*rows    */ rowsB,
                /*cols    */ cols,
                /*act     */ act,
                /*alpha   */ alpha,
                /*gelu    */ gelu_tanh_flag,
                /*stream  */ stream
            );
            CUDA_CHECK(hipGetLastError());
            break;
        }

        // -------- Softmax: 필요 시 일반 backward (fused면 위에서 스킵됨) --------
        case SOFTMAX:
        {
            // fused가 아니면 일반 softmax backward 수행
            const int rowsB = batch_size * out_shape.rows;
            const int cols  = out_shape.cols;

            const float* gout = grad_out_full;           // dL/dY
            const float* y    = tensors[op.output_id];   // Y = softmax(X)
            float* gin        = grad_input_full;         // dL/dX

            float temperature = (op.extra_params.temperature > 0.f)
                              ? op.extra_params.temperature : 1.f;
            hipStream_t stream = 0;

            launch_softmax_backward(
                /*grad_out*/ gout,
                /*out     */ y,
                /*grad_in */ gin,
                /*rows    */ rowsB,
                /*cols    */ cols,
                /*temperature*/ temperature,
                /*stream  */ stream
            );
            CUDA_CHECK(hipGetLastError());
            break;
        }

        case FLATTEN: {
            // shape만 바뀌는 op → 그래디언트 패스-스루
            gradients[op.input_id] = grad_out_full;
            continue;
        }

        default:
            // 다른 OpType은 위에서 처리됨
            break;
        }

        if (op.op_type != FLATTEN) {
            if (!grad_input_full) {
                std::fprintf(stderr, "[BW] grad_input_full is null: op=%d\n", op.op_type);
            } else {
                gradients[op.input_id] = grad_input_full;
            }
        }
    }
}
