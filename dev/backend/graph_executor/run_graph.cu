#include "hip/hip_runtime.h"
// run_graph.cu (updated: cuBLAS Strided-Batched + bias fuse + single-launch activations)

#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <unordered_map>
#include <hip/hip_runtime.h>

#include <hipblas.h>

#include "quant/quant_types.cuh"
#include "quant/observers.cuh"
#include "quant/quant_kernels.cuh"
#include "quant/int8_gemm_dp4a.cuh"
#include "quant/epilogue_kernels.cuh"

#include "run_graph.cuh"
#include "activation_ops.cuh"
#include "softmax_kernels.cuh"
#include "add_bias_rowwise.cuh"
#include "cnn_kernels.cuh"
#include "op_structs.cuh"

#include "ge/cuda_check.cuh"
#include "ge/cublas_utils.cuh"
#include "ge/gemm_rm.cuh"
#include "ge/act_map.cuh"
#include "ge/alloc_utils.cuh"

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif


static void debug_l2(const char* name, const float* dptr, size_t n_elems) {
    std::vector<float> h(n_elems);
    hipMemcpy(h.data(), dptr, n_elems * sizeof(float), hipMemcpyDeviceToHost);
    double s = 0.0;
    for (size_t i = 0; i < n_elems; ++i) { double v = h[i]; s += v * v; }
    std::fprintf(stderr, "[GRAD] %s L2=%.6e\n", name, std::sqrt(s));
}


// 입력 텐서 조회
inline const float* get_tensor_ro(const std::unordered_map<std::string, float*>& tensors,
                                  const std::string& id)
{
    auto it = tensors.find(id);
    if (it == tensors.end() || !it->second) {
        std::fprintf(stderr, "[ERR] Tensor '%s' not found (RO)\n", id.c_str());
        return nullptr;
    }
    return it->second;
}

inline float* get_tensor_rw(std::unordered_map<std::string, float*>& tensors,
                            const std::string& id)
{
    auto it = tensors.find(id);
    if (it == tensors.end() || !it->second) {
        std::fprintf(stderr, "[ERR] Tensor '%s' not found (RW)\n", id.c_str());
        return nullptr;
    }
    return it->second;
}

void run_graph_cuda(
    const std::vector<OpStruct>& E,
    std::unordered_map<std::string, float*>& tensors,
    std::unordered_map<std::string, Shape>& shapes,
    float* out_host,
    const std::string& final_output_id,
    int batch_size)
{
    auto h = ge_cublas();

    for (size_t i = 0; i < E.size(); ++i) {
        const auto& op = E[i];
        if (op.op_type == LOSS) continue;

        // 입력/shape 확보
        auto it_in = tensors.find(op.input_id);
        if (it_in == tensors.end() || it_in->second == nullptr) {
            std::fprintf(stderr, "[ERR] missing input tensor: %s\n", op.input_id.c_str());
            break;
        }
        float* input = it_in->second;

        const auto it_inshape = shapes.find(op.input_id);
        if (it_inshape == shapes.end()) {
            std::fprintf(stderr, "[ERR] missing input shape: %s\n", op.input_id.c_str());
            break;
        }
        const Shape in_shape = it_inshape->second;

        float* param = nullptr;
        if (!op.param_id.empty()) {
            auto it_p = tensors.find(op.param_id);
            if (it_p != tensors.end()) param = it_p->second;
        }

        Shape out_shape = in_shape; // 기본은 동일

        switch (op.op_type) {
        case MATMUL: {
            if (!param) {
                std::fprintf(stderr, "[MATMUL] missing param for %s\n", op.output_id.c_str());
                break;
            }
            auto it_wshape = shapes.find(op.param_id);
            if (it_wshape == shapes.end()) {
                std::fprintf(stderr, "[MATMUL] missing weight shape: %s\n", op.param_id.c_str());
                break;
            }
            const Shape w_shape = it_wshape->second; // [K, N]
            const int M = in_shape.rows;
            const int K = in_shape.cols;
            const int N = w_shape.cols;
            if (w_shape.rows != K) {
                std::fprintf(stderr, "[MATMUL] dim mismatch: in(K=%d) vs W(rows=%d)\n", K, w_shape.rows);
                break;
            }
            out_shape = { M, N };

            // 다음 op가 row-wise ADD면 bias fuse
            bool fuse_bias = false;
            float* bias_ptr = nullptr;
            std::string out_id = op.output_id;

            if ((i + 1) < E.size()) {
                const auto& nx = E[i + 1];
                if (nx.op_type == ADD && nx.input_id == op.output_id &&
                    !nx.param_id.empty() && tensors.count(nx.param_id))
                {
                    const Shape bshape = shapes[nx.param_id];
                    const bool row_bias = (bshape.rows == 1 && bshape.cols == N) ||
                                          (bshape.rows == N && bshape.cols == 1);
                    if (row_bias) {
                        fuse_bias = true;
                        bias_ptr = tensors[nx.param_id];
                        out_id = nx.output_id; // ADD 출력으로 바로 기록
                    }
                }
            }

            float* Y = ge_ensure_output(tensors, shapes, out_id, out_shape, batch_size);

            const long long strideA = (long long)M * K;
            const long long strideC = (long long)M * N;

            gemm_rm_strided_batched_tf32(
                h,
                /*transA=*/false, /*transB=*/false,
                /*M=*/M, /*N=*/N, /*K=*/K,
                /*A*/ input,  /*lda=*/K, /*strideA=*/strideA,
                /*B*/ param,  /*ldb=*/N, /*strideB=*/0LL,   // shared weight
                /*C*/ Y,      /*ldc=*/N, /*strideC=*/strideC,
                /*batch=*/batch_size,
                /*alpha=*/1.f, /*beta=*/0.f
            );

            if (fuse_bias) {
                const int rowsB = batch_size * M;
                const int cols  = N;
                launch_add_bias_rowwise(Y, bias_ptr, Y, rowsB, cols);
                CUDA_CHECK(hipGetLastError());
                ++i; // 다음 ADD 스킵
            }
            break;
        }

        case ADD: {
            if (!param) {
                std::fprintf(stderr, "[ADD] missing param for %s\n", op.output_id.c_str());
                break;
            }
            out_shape = in_shape;
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const Shape bshape = shapes[op.param_id];

            const int rows_per_sample = out_shape.rows; // 예: filters(채널)
            const int cols            = out_shape.cols; // 예: H*W
            const int rowsB           = batch_size * rows_per_sample;

            const bool bias_rowwise = (bshape.rows == 1 && bshape.cols == cols)   // (1, cols)
                                || (bshape.rows == cols && bshape.cols == 1);  // (cols, 1)

            const bool bias_colwise = (bshape.rows == 1 && bshape.cols == rows_per_sample)   // (1, rows)
                                || (bshape.rows == rows_per_sample && bshape.cols == 1);  // (rows, 1)

            if (bias_rowwise) {
                launch_add_bias_rowwise(input, param, output, rowsB, cols);  // stream 없음 오버로드
                CUDA_CHECK(hipGetLastError());
            } else if (bias_colwise) {
                launch_add_bias_colwise(input, param, output, rowsB, cols, rows_per_sample);
                CUDA_CHECK(hipGetLastError());
            } else {
                std::fprintf(stderr,
                    "[ADD] unsupported shape: input(%d,%d) + param(%d,%d). "
                    "Expect row-wise (len=cols) or channel-wise (len=rows) bias.\n",
                    in_shape.rows, in_shape.cols, bshape.rows, bshape.cols);
                const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
                CUDA_CHECK(hipMemcpy(output, input, bytes, hipMemcpyDeviceToDevice));
            }
            break;
        }

        // ---------- 활성화 ----------
        case SIGMOID:
        case RELU:
        case TANH:
        case LEAKY_RELU:
        case ELU:
        case GELU:
        case SILU: {
            out_shape = (shapes.count(op.output_id) ? shapes[op.output_id] : in_shape);
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const int rowsB = batch_size * out_shape.rows;
            const int cols  = out_shape.cols;

            const float* bias_ptr = nullptr; // 옵션
            if (!op.param_id.empty()) {
                auto itb = tensors.find(op.param_id);
                if (itb != tensors.end()) bias_ptr = itb->second;
            }
            const int act = ge_map_act_type(op.op_type);
            const float alpha = op.extra_params.alpha;
            const int gelu_tanh_flag = op.extra_params.gelu_tanh ? 1 : 0;

            launch_activation_forward(input, bias_ptr, output,
                                      rowsB, cols, act, alpha, gelu_tanh_flag, /*stream*/0);
            CUDA_CHECK(hipGetLastError());
            break;
        }

        // ---------- Softmax ----------
        case SOFTMAX: {
            out_shape = (shapes.count(op.output_id) ? shapes[op.output_id] : in_shape);
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const int rowsB = batch_size * out_shape.rows;
            const int cols  = out_shape.cols;
            const float temperature = (op.extra_params.temperature > 0.f)
                                        ? op.extra_params.temperature : 1.f;

            launch_softmax_forward(input, output, rowsB, cols, temperature, /*stream*/0);
            CUDA_CHECK(hipGetLastError());
            break;
        }

        case FLATTEN: {
            auto it_outshape = shapes.find(op.output_id);
            if (it_outshape == shapes.end()) {
                std::fprintf(stderr, "[FLATTEN][ERR] missing out shape for %s\n", op.output_id.c_str());
                break;
            }
            const Shape out_shape_f = it_outshape->second;
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape_f, batch_size);

            const size_t elems_in  = (size_t)batch_size * in_shape.rows      * in_shape.cols;
            const size_t elems_out = (size_t)batch_size * out_shape_f.rows    * out_shape_f.cols;
            if (elems_in != elems_out) {
                std::fprintf(stderr,
                        "[FLATTEN][ERR] elem mismatch: in=%zu out=%zu (B=%d in=(%d,%d) out=(%d,%d))\n",
                        elems_in, elems_out, batch_size,
                        in_shape.rows, in_shape.cols, out_shape_f.rows, out_shape_f.cols);
                break;
            }
            CUDA_CHECK(hipMemcpy(output, input, elems_in * sizeof(float), hipMemcpyDeviceToDevice));
            break;
        }

        case CONV2D: {
            const OpExtraParams& ex = op.extra_params;

            const int B    = batch_size;
            const int Cin  = ex.input_c;
            const int Hin  = ex.input_h;
            const int Win  = ex.input_w;
            const int Cout = ex.output_c;
            const int Kh   = ex.kernel_h;
            const int Kw   = ex.kernel_w;
            const int Sh   = (ex.stride_h > 0 ? ex.stride_h : 1);
            const int Sw   = (ex.stride_w > 0 ? ex.stride_w : 1);
            const int Ph   = ex.padding_h;
            const int Pw   = ex.padding_w;

            const int Hout = (Hin + 2*Ph - Kh) / Sh + 1;
            const int Wout = (Win + 2*Pw - Kw) / Sw + 1;

            // 출력 Shape = (rows=Cout, cols=Hout*Wout)
            Shape out_shape_c{Cout, Hout * Wout};
            shapes[op.output_id] = out_shape_c;

            float* X = tensors[op.input_id];  // [B,Hin,Win,Cin]
            float* W = tensors[op.param_id];  // [Cout,Cin,Kh,Kw] (연속)
            float* Y = ge_ensure_output(tensors, shapes, op.output_id, out_shape_c, B);

            launch_conv2d_forward_nhwc(
                /*X=*/X, /*W=*/W, /*Y=*/Y,
                /*B=*/B, /*Hin=*/Hin, /*Win=*/Win, /*Cin=*/Cin,
                /*Hout=*/Hout, /*Wout=*/Wout, /*Cout=*/Cout,
                /*Kh=*/Kh, /*Kw=*/Kw, /*Sh=*/Sh, /*Sw=*/Sw, /*Ph=*/Ph, /*Pw=*/Pw,
                /*stream=*/0);
            CUDA_CHECK(hipGetLastError());
            break;
        }


        default:
            std::fprintf(stderr, "[ERR] Unsupported op_type: %d\n", op.op_type);
            break;
        } // switch
    } // for

    // 최종 출력 호스트 복사
    auto it_final  = tensors.find(final_output_id);
    auto it_fshape = shapes.find(final_output_id);
    if (out_host && it_final != tensors.end() && it_fshape != shapes.end()) {
        const Shape out_shape = it_fshape->second;
        const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
        CUDA_CHECK(hipMemcpy(out_host, it_final->second, bytes, hipMemcpyDeviceToHost));
    } else if (out_host) {
        std::fprintf(stderr, "[ERR] final output missing: id=%s\n", final_output_id.c_str());
    }
}
