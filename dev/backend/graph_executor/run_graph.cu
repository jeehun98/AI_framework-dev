// run_graph.cu (updated: cuBLAS Strided-Batched + bias fuse + single-launch activations)

#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <unordered_map>
#include <hip/hip_runtime.h>

#include <hipblas.h>

#include "quant/quant_types.cuh"
#include "quant/observers.cuh"
#include "quant/quant_kernels.cuh"
#include "quant/int8_gemm_dp4a.cuh"
#include "quant/epilogue_kernels.cuh"

#include "run_graph.cuh"
#include "activation_ops.cuh"
#include "softmax_kernels.cuh"
#include "add_bias_rowwise.cuh"
#include "cnn_kernels.cuh"
#include "op_structs.cuh"

#include "ge/cuda_check.cuh"
#include "ge/cublas_utils.cuh"
#include "ge/gemm_rm.cuh"
#include "ge/act_map.cuh"
#include "ge/alloc_utils.cuh"

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

// 입력 텐서 조회
inline const float* get_tensor_ro(const std::unordered_map<std::string, float*>& tensors,
                                  const std::string& id)
{
    auto it = tensors.find(id);
    if (it == tensors.end() || !it->second) {
        std::fprintf(stderr, "[ERR] Tensor '%s' not found (RO)\n", id.c_str());
        return nullptr;
    }
    return it->second;
}

inline float* get_tensor_rw(std::unordered_map<std::string, float*>& tensors,
                            const std::string& id)
{
    auto it = tensors.find(id);
    if (it == tensors.end() || !it->second) {
        std::fprintf(stderr, "[ERR] Tensor '%s' not found (RW)\n", id.c_str());
        return nullptr;
    }
    return it->second;
}

void run_graph_cuda(
    const std::vector<OpStruct>& E,
    std::unordered_map<std::string, float*>& tensors,
    std::unordered_map<std::string, Shape>& shapes,
    float* out_host,
    const std::string& final_output_id,
    int batch_size)
{
    auto h = ge_cublas();

    for (size_t i = 0; i < E.size(); ++i) {
        const auto& op = E[i];
        if (op.op_type == LOSS) continue;

        // 입력/shape 확보
        auto it_in = tensors.find(op.input_id);
        if (it_in == tensors.end() || it_in->second == nullptr) {
            std::fprintf(stderr, "[ERR] missing input tensor: %s\n", op.input_id.c_str());
            break;
        }
        float* input = it_in->second;

        const auto it_inshape = shapes.find(op.input_id);
        if (it_inshape == shapes.end()) {
            std::fprintf(stderr, "[ERR] missing input shape: %s\n", op.input_id.c_str());
            break;
        }
        const Shape in_shape = it_inshape->second;

        float* param = nullptr;
        if (!op.param_id.empty()) {
            auto it_p = tensors.find(op.param_id);
            if (it_p != tensors.end()) param = it_p->second;
        }

        Shape out_shape = in_shape; // 기본은 동일

        switch (op.op_type) {
        case MATMUL: {
            if (!param) {
                std::fprintf(stderr, "[MATMUL] missing param for %s\n", op.output_id.c_str());
                break;
            }
            auto it_wshape = shapes.find(op.param_id);
            if (it_wshape == shapes.end()) {
                std::fprintf(stderr, "[MATMUL] missing weight shape: %s\n", op.param_id.c_str());
                break;
            }
            const Shape w_shape = it_wshape->second; // [K, N]
            const int M = in_shape.rows;
            const int K = in_shape.cols;
            const int N = w_shape.cols;
            if (w_shape.rows != K) {
                std::fprintf(stderr, "[MATMUL] dim mismatch: in(K=%d) vs W(rows=%d)\n", K, w_shape.rows);
                break;
            }
            out_shape = { M, N };

            // 다음 op가 row-wise ADD면 bias fuse
            bool fuse_bias = false;
            float* bias_ptr = nullptr;
            std::string out_id = op.output_id;

            if ((i + 1) < E.size()) {
                const auto& nx = E[i + 1];
                if (nx.op_type == ADD && nx.input_id == op.output_id &&
                    !nx.param_id.empty() && tensors.count(nx.param_id))
                {
                    const Shape bshape = shapes[nx.param_id];
                    const bool row_bias = (bshape.rows == 1 && bshape.cols == N) ||
                                          (bshape.rows == N && bshape.cols == 1);
                    if (row_bias) {
                        fuse_bias = true;
                        bias_ptr = tensors[nx.param_id];
                        out_id = nx.output_id; // ADD 출력으로 바로 기록
                    }
                }
            }

            float* Y = ge_ensure_output(tensors, shapes, out_id, out_shape, batch_size);

            const long long strideA = (long long)M * K;
            const long long strideC = (long long)M * N;

            gemm_rm_strided_batched_tf32(
                h,
                /*transA=*/false, /*transB=*/false,
                /*M=*/M, /*N=*/N, /*K=*/K,
                /*A*/ input,  /*lda=*/K, /*strideA=*/strideA,
                /*B*/ param,  /*ldb=*/N, /*strideB=*/0LL,   // shared weight
                /*C*/ Y,      /*ldc=*/N, /*strideC=*/strideC,
                /*batch=*/batch_size,
                /*alpha=*/1.f, /*beta=*/0.f
            );

            if (fuse_bias) {
                const int rowsB = batch_size * M;
                const int cols  = N;
                launch_add_bias_rowwise(Y, bias_ptr, Y, rowsB, cols);
                CUDA_CHECK(hipGetLastError());
                ++i; // 다음 ADD 스킵
            }
            break;
        }

        case ADD: {
            if (!param) {
                std::fprintf(stderr, "[ADD] missing param for %s\n", op.output_id.c_str());
                break;
            }
            out_shape = in_shape;
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const Shape bshape = shapes[op.param_id];
            const bool row_bias = (bshape.rows == 1 && bshape.cols == out_shape.cols) ||
                                  (bshape.rows == out_shape.cols && bshape.cols == 1);
            if (row_bias) {
                const int rowsB = batch_size * out_shape.rows;
                const int cols  = out_shape.cols;
                launch_add_bias_rowwise(input, param, output, rowsB, cols);
                CUDA_CHECK(hipGetLastError());
            } else {
                std::fprintf(stderr,
                        "[ADD] unsupported shape: input(%d,%d) + param(%d,%d). Expect row-wise bias.\n",
                        in_shape.rows, in_shape.cols, bshape.rows, bshape.cols);
                const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
                CUDA_CHECK(hipMemcpy(output, input, bytes, hipMemcpyDeviceToDevice));
            }
            break;
        }

        // ---------- 활성화 ----------
        case SIGMOID:
        case RELU:
        case TANH:
        case LEAKY_RELU:
        case ELU:
        case GELU:
        case SILU: {
            out_shape = (shapes.count(op.output_id) ? shapes[op.output_id] : in_shape);
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const int rowsB = batch_size * out_shape.rows;
            const int cols  = out_shape.cols;

            const float* bias_ptr = nullptr; // 옵션
            if (!op.param_id.empty()) {
                auto itb = tensors.find(op.param_id);
                if (itb != tensors.end()) bias_ptr = itb->second;
            }
            const int act = ge_map_act_type(op.op_type);
            const float alpha = op.extra_params.alpha;
            const int gelu_tanh_flag = op.extra_params.gelu_tanh ? 1 : 0;

            launch_activation_forward(input, bias_ptr, output,
                                      rowsB, cols, act, alpha, gelu_tanh_flag, /*stream*/0);
            CUDA_CHECK(hipGetLastError());
            break;
        }

        // ---------- Softmax ----------
        case SOFTMAX: {
            out_shape = (shapes.count(op.output_id) ? shapes[op.output_id] : in_shape);
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const int rowsB = batch_size * out_shape.rows;
            const int cols  = out_shape.cols;
            const float temperature = (op.extra_params.temperature > 0.f)
                                        ? op.extra_params.temperature : 1.f;

            launch_softmax_forward(input, output, rowsB, cols, temperature, /*stream*/0);
            CUDA_CHECK(hipGetLastError());
            break;
        }

        case FLATTEN: {
            auto it_outshape = shapes.find(op.output_id);
            if (it_outshape == shapes.end()) {
                std::fprintf(stderr, "[FLATTEN][ERR] missing out shape for %s\n", op.output_id.c_str());
                break;
            }
            const Shape out_shape_f = it_outshape->second;
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape_f, batch_size);

            const size_t elems_in  = (size_t)batch_size * in_shape.rows      * in_shape.cols;
            const size_t elems_out = (size_t)batch_size * out_shape_f.rows    * out_shape_f.cols;
            if (elems_in != elems_out) {
                std::fprintf(stderr,
                        "[FLATTEN][ERR] elem mismatch: in=%zu out=%zu (B=%d in=(%d,%d) out=(%d,%d))\n",
                        elems_in, elems_out, batch_size,
                        in_shape.rows, in_shape.cols, out_shape_f.rows, out_shape_f.cols);
                break;
            }
            CUDA_CHECK(hipMemcpy(output, input, elems_in * sizeof(float), hipMemcpyDeviceToDevice));
            break;
        }

        case CONV2D: {
            auto it_outshape = shapes.find(op.output_id);
            if (it_outshape == shapes.end()) {
                std::fprintf(stderr, "[CONV2D][ERR] missing out shape for %s\n", op.output_id.c_str());
                break;
            }
            const Shape out_shape_c = it_outshape->second;
            float* output = ge_ensure_output(tensors, shapes, op.output_id, out_shape_c, batch_size);

            const int KH = op.extra_params.kernel_h;
            const int KW = op.extra_params.kernel_w;
            const int IH = op.extra_params.input_h;
            const int IW = op.extra_params.input_w;
            const int IC = op.extra_params.input_c;
            const int OC = op.extra_params.output_c;

            const int OW = out_shape_c.cols / OC;
            const int OH = out_shape_c.rows;

            dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
            dim3 gridDim((OW + TILE_WIDTH - 1) / TILE_WIDTH,
                         (OH + TILE_WIDTH - 1) / TILE_WIDTH,
                         OC);

            for (int b = 0; b < batch_size; ++b) {
                float* in_b  = input  + (size_t)b * IH * IW * IC;
                float* out_b = output + (size_t)b * OH * OW * OC;
                conv2d_forward_kernel<<<gridDim, blockDim>>>(
                    in_b, param, out_b,
                    /*batch_size=*/1, IH, IW,
                    IC, OC,
                    KH, KW,
                    OH, OW
                );
                CUDA_CHECK(hipGetLastError());
            }
            break;
        }

        default:
            std::fprintf(stderr, "[ERR] Unsupported op_type: %d\n", op.op_type);
            break;
        } // switch
    } // for

    // 최종 출력 호스트 복사
    auto it_final  = tensors.find(final_output_id);
    auto it_fshape = shapes.find(final_output_id);
    if (out_host && it_final != tensors.end() && it_fshape != shapes.end()) {
        const Shape out_shape = it_fshape->second;
        const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
        CUDA_CHECK(hipMemcpy(out_host, it_final->second, bytes, hipMemcpyDeviceToHost));
    } else if (out_host) {
        std::fprintf(stderr, "[ERR] final output missing: id=%s\n", final_output_id.c_str());
    }
}
