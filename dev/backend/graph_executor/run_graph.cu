// run_graph.cu (updated: cuBLAS Strided-Batched + bias fuse + single-launch activations)
#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#include <unordered_map>
#include <hipblas.h>

#include "run_graph.cuh"
#include "activation_ops.cuh"
#include "softmax_kernels.cuh"
#include "add_bias_rowwise.cuh"
#include "cnn_kernels.cuh"
#include "op_structs.cuh"

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

#ifndef CUDA_CHECK
#define CUDA_CHECK(x) do { hipError_t _e=(x); if(_e!=hipSuccess){ \
  fprintf(stderr,"[CUDA] %s:%d %s\n", __FILE__, __LINE__, hipGetErrorString(_e)); } } while(0)
#endif

// === cuBLAS 에러 체크 ===
#ifndef CUBLAS_CHECK
#define CUBLAS_CHECK(call) do { \
    hipblasStatus_t _st = (call); \
    if (_st != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "[cuBLAS] %s:%d status=%d\n", __FILE__, __LINE__, (int)_st); \
    } \
} while(0)
#endif

// === 전역 cuBLAS 핸들 재사용 ===
static hipblasHandle_t g_cublas = nullptr;

static void ensure_cublas() {
    if (!g_cublas) {
        CUBLAS_CHECK(hipblasCreate(&g_cublas));
        // Ampere+에서 TF32 쓰고 싶으면 다음 줄 주석 해제
        // hipblasSetMathMode(g_cublas, HIPBLAS_TF32_TENSOR_OP_MATH);
    }
}

// ------------------- 텐서/그래디언트 헬퍼 -------------------
inline const float* get_tensor_ptr(
    const std::unordered_map<std::string, uintptr_t>& tensors,
    const std::string& id)
{
    auto it = tensors.find(id);
    if (it == tensors.end()) {
        fprintf(stderr, "[ERROR] Tensor ID '%s' not found (get_tensor_ptr)\n", id.c_str());
        return nullptr;
    }
    return reinterpret_cast<const float*>(it->second);
}

inline float* get_tensor_ptr_rw(
    std::unordered_map<std::string, uintptr_t>& tensors,
    const std::string& id)
{
    auto it = tensors.find(id);
    if (it == tensors.end()) {
        fprintf(stderr, "[ERROR] Tensor ID '%s' not found (get_tensor_ptr_rw)\n", id.c_str());
        return nullptr;
    }
    return reinterpret_cast<float*>(it->second);
}

// 필요 시 출력 텐서를 새로 할당해 등록
inline float* ensure_output(std::unordered_map<std::string, uintptr_t>& tensors,
                            const std::unordered_map<std::string, Shape>& shapes,
                            const std::string& out_id,
                            const Shape& out_shape,
                            int batch_size)
{
    auto it = tensors.find(out_id);
    if (it != tensors.end()) {
        return reinterpret_cast<float*>(it->second);
    }
    size_t elems = static_cast<size_t>(batch_size) * out_shape.rows * out_shape.cols;
    float* dptr = nullptr;
    CUDA_CHECK(hipMalloc(&dptr, elems * sizeof(float)));
    tensors[out_id] = reinterpret_cast<uintptr_t>(dptr);
    return dptr;
}

// ------------------- 활성화 매핑 -------------------
static inline int map_act_type(int op_type) {
    switch (op_type) {
        case SIGMOID:    return ACT_SIGMOID;
        case RELU:       return ACT_RELU;
        case TANH:       return ACT_TANH;
        case LEAKY_RELU: return ACT_LEAKY;
        case ELU:        return ACT_ELU;
        case GELU:       return ACT_GELU;
        case SILU:       return ACT_SILU;
        default:         return ACT_IDENTITY;
    }
}

// 외부에서 스트림을 관리한다면 그대로 넘기고, 없다면 0 사용 가능
static inline hipStream_t pick_stream(hipStream_t user_stream) {
    return user_stream; // 필요 시 nullptr/0 허용
}

// -----------------------------------------------------------------------------
// row-major + StridedBatched (A/B/C가 등간격 스트라이드로 배치 반복) - FP32 기본
static inline void gemm_rm_strided_batched(
    hipblasHandle_t h,
    bool transA, bool transB,
    int M, int N, int K,
    const float* A, int lda, long long strideA,
    const float* B, int ldb, long long strideB,
    float* C, int ldc, long long strideC,
    int batch,
    float alpha=1.f, float beta=0.f)
{
    hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    CUBLAS_CHECK(
        hipblasSgemmStridedBatched(
            h, opB, opA, N, M, K,
            &alpha,
            B, ldb, strideB,
            A, lda, strideA,
            &beta,
            C, ldc, strideC,
            batch
        )
    );
}

// row-major + StridedBatched (TF32 fast) - 권장 경로
static inline void gemm_rm_strided_batched_tf32(
    hipblasHandle_t h,
    bool transA, bool transB,
    int M, int N, int K,
    const float* A, int lda, long long strideA,
    const float* B, int ldb, long long strideB,
    float* C, int ldc, long long strideC,
    int batch,
    float alpha=1.f, float beta=0.f)
{
    hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    CUBLAS_CHECK(
        hipblasGemmStridedBatchedEx(
            h,
            /*opB,opA*/ opB, opA,
            /*m,n,k*/   N,   M,   K,
            &alpha,
            /*B*/ B, HIP_R_32F, ldb, strideB,
            /*A*/ A, HIP_R_32F, lda, strideA,
            &beta,
            /*C*/ C, HIP_R_32F, ldc, strideC,
            /*batch*/ batch,
            /*computeType*/ HIPBLAS_COMPUTE_32F_FAST_TF32,
            /*algo*/ CUBLAS_GEMM_DEFAULT_TENSOR_OP
        )
    );
}
// -----------------------------------------------------------------------------

static inline float* ensure_output(std::unordered_map<std::string, float*>& tensors,
                                   std::unordered_map<std::string, Shape>& shapes,
                                   const std::string& out_id,
                                   const Shape& out_shape,
                                   int batch_size)
{
    auto it = tensors.find(out_id);
    if (it != tensors.end()) {
        shapes[out_id] = out_shape;
        return it->second;
    }
    float* out_ptr = nullptr;
    const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
    CUDA_CHECK(hipMalloc(&out_ptr, bytes));
    tensors[out_id] = out_ptr;
    shapes[out_id] = out_shape;
    return out_ptr;
}

void run_graph_cuda(
    const std::vector<OpStruct>& E,
    std::unordered_map<std::string, float*>& tensors,
    std::unordered_map<std::string, Shape>& shapes,
    float* out_host,
    const std::string& final_output_id,
    int batch_size)
{
    for (size_t i = 0; i < E.size(); ++i) {
        const auto& op = E[i];
        if (op.op_type == LOSS) continue;

        float* input = tensors[op.input_id];
        float* param = (!op.param_id.empty() && tensors.find(op.param_id) != tensors.end())
                         ? tensors[op.param_id] : nullptr;

        const Shape in_shape = shapes[op.input_id];
        Shape out_shape = in_shape;

        switch (op.op_type) {
        case MATMUL: {
            if (!param) {
                fprintf(stderr, "[MATMUL] missing param for %s\n", op.output_id.c_str());
                break;
            }
            // A[M,K] * W[K,N] = C[M,N]
            const Shape w_shape = shapes[op.param_id]; // [K, N]
            const int M = in_shape.rows;
            const int K = in_shape.cols;
            const int N = w_shape.cols;
            out_shape = { M, N };

            // 다음 op가 row-wise ADD면 bias fuse
            bool fuse_bias = false;
            float* bias_ptr = nullptr;
            std::string out_id = op.output_id;

            if ((i + 1) < E.size()) {
                const auto& next = E[i + 1];
                if (next.op_type == ADD && next.input_id == op.output_id &&
                    !next.param_id.empty() && tensors.count(next.param_id))
                {
                    const Shape bshape = shapes[next.param_id];
                    const bool row_bias = (bshape.rows == 1 && bshape.cols == N) ||
                                          (bshape.rows == N && bshape.cols == 1);
                    if (row_bias) {
                        fuse_bias = true;
                        bias_ptr = tensors[next.param_id];
                        out_id = next.output_id; // ADD 출력으로 바로 기록
                    }
                }
            }

            float* Y = ensure_output(tensors, shapes, out_id, out_shape, batch_size);

            // 배치 루프 없이 GEMM 1회
            ensure_cublas();
            const long long strideA = (long long)M * K;
            const long long strideC = (long long)M * N;

            // TF32 경로
            gemm_rm_strided_batched_tf32(
                g_cublas,
                /*transA=*/false, /*transB=*/false,
                /*M=*/M, /*N=*/N, /*K=*/K,
                /*A =*/ input,              /*lda =*/ K, /*strideA =*/ strideA,
                /*B =*/ param,              /*ldb =*/ N, /*strideB =*/ 0LL, // 공유 가중치
                /*C =*/ Y,                  /*ldc =*/ N, /*strideC =*/ strideC,
                /*batch=*/batch_size,
                /*alpha=*/1.f, /*beta=*/0.f
            );

            // bias를 한 번에 더함 (ADD fuse)
            if (fuse_bias) {
                const int rowsB = batch_size * M;
                const int cols  = N;
                launch_add_bias_rowwise(/*input=*/Y, /*bias=*/bias_ptr, /*output=*/Y,
                                        /*rows=*/rowsB, /*cols=*/cols);
                CUDA_CHECK(hipGetLastError());
                ++i; // 다음 ADD 스킵
            }
            break;
        }

        case ADD: {
            if (!param) {
                fprintf(stderr, "[ADD] missing param for %s\n", op.output_id.c_str());
                break;
            }
            out_shape = in_shape;
            float* output = ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);

            const Shape bshape = shapes[op.param_id];
            const bool row_bias = (bshape.rows == 1 && bshape.cols == out_shape.cols) ||
                                  (bshape.rows == out_shape.cols && bshape.cols == 1);

            if (row_bias) {
                // 배치까지 합쳐 한 번에
                const int rowsB = batch_size * out_shape.rows;
                const int cols  = out_shape.cols;
                launch_add_bias_rowwise(/*input=*/input, /*bias=*/param, /*output=*/output,
                                        /*rows=*/rowsB, /*cols=*/cols);
                CUDA_CHECK(hipGetLastError());
            } else {
                // 필요시 다른 add 구현 추가
                const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
                CUDA_CHECK(hipMemcpy(output, input, bytes, hipMemcpyDeviceToDevice));
                fprintf(stderr,
                        "[ADD] unsupported shape: input(%d,%d) + param(%d,%d). Expect row-wise bias.\n",
                        in_shape.rows, in_shape.cols, bshape.rows, bshape.cols);
            }
            break;
        }

        // ---------- 활성화( bias 선택 지원 ) ----------
        case SIGMOID:
        case RELU:
        case TANH:
        case LEAKY_RELU:
        case ELU:
        case GELU:
        case SILU:
        {
            Shape act_shape = shapes[op.output_id];
            float* output   = ensure_output(tensors, shapes, op.output_id, act_shape, batch_size);

            const int rowsB = batch_size * act_shape.rows;
            const int cols  = act_shape.cols;

            const float* in_ptr   = input;      // pre-activation z
            const float* bias_ptr = nullptr;    // 선택적 bias
            if (!op.param_id.empty()) {
                auto it = tensors.find(op.param_id);
                if (it != tensors.end()) bias_ptr = it->second;
            }

            const int act = map_act_type(op.op_type);
            const float alpha = op.extra_params.alpha;            // Leaky/ELU
            const int gelu_tanh_flag = op.extra_params.gelu_tanh ? 1 : 0;

            hipStream_t stream = 0; // 이 함수 서명엔 stream이 없으므로 기본 스트림 사용

            launch_activation_forward(
                /*in*/   in_ptr,
                /*bias*/ bias_ptr,
                /*out*/  output,
                /*rows*/ rowsB,
                /*cols*/ cols,
                /*act*/  act,
                /*alpha*/alpha,
                /*gelu_tanh*/ gelu_tanh_flag,
                /*stream*/ stream
            );
            CUDA_CHECK(hipGetLastError());
            break;
        }

        // ---------- Softmax(행 기준, temperature 지원) ----------
        case SOFTMAX:
        {
            Shape sm_shape = shapes[op.output_id];
            float* output  = ensure_output(tensors, shapes, op.output_id, sm_shape, batch_size);
            const float* in_ptr = input;

            const int rowsB = batch_size * sm_shape.rows;
            const int cols  = sm_shape.cols;

            float temperature = (op.extra_params.temperature > 0.f)
                                  ? op.extra_params.temperature : 1.f;
            hipStream_t stream = 0;

            launch_softmax_forward(
                /*in*/   in_ptr,
                /*out*/  output,
                /*rows*/ rowsB,
                /*cols*/ cols,
                /*temperature*/ temperature,
                /*stream*/ stream
            );
            CUDA_CHECK(hipGetLastError());
            break;
        }

        case FLATTEN: {
            // 통짜 D2D 복사
            float* output = ensure_output(tensors, shapes, op.output_id, out_shape, batch_size);
            const size_t bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
            CUDA_CHECK(hipMemcpy(output, input, bytes, hipMemcpyDeviceToDevice));
            break;
        }

        case CONV2D: {
            // (기존 구현 유지)
            int KH = op.extra_params.kernel_h;
            int KW = op.extra_params.kernel_w;
            int SH = op.extra_params.stride_h;
            int SW = op.extra_params.stride_w;
            int PH = op.extra_params.padding_h;
            int PW = op.extra_params.padding_w;
            int IH = op.extra_params.input_h;
            int IW = op.extra_params.input_w;
            int IC = op.extra_params.input_c;
            int OC = op.extra_params.output_c;

            const int OW = shapes[op.output_id].cols / OC;
            const int OH = shapes[op.output_id].rows;

            float* output = ensure_output(tensors, shapes, op.output_id, shapes[op.output_id], batch_size);

            dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
            dim3 gridDim((OW + TILE_WIDTH - 1) / TILE_WIDTH,
                         (OH + TILE_WIDTH - 1) / TILE_WIDTH,
                         OC);

            for (int b = 0; b < batch_size; ++b) {
                float* in_b  = input  + (size_t)b * IH * IW * IC;
                float* out_b = output + (size_t)b * OH * OW * OC;
                conv2d_forward_kernel<<<gridDim, blockDim>>>(
                    in_b, param, out_b,
                    /*batch_size=*/1, IH, IW,
                    IC, OC,
                    KH, KW,
                    OH, OW
                );
                CUDA_CHECK(hipGetLastError());
            }
            break;
        }

        default:
            fprintf(stderr, "[ERROR] Unsupported op_type: %d\n", op.op_type);
            break;
        } // switch
    } // for i

    // 최종 출력 호스트로 복사
    const Shape out_shape = shapes[final_output_id];
    const size_t out_bytes = (size_t)batch_size * out_shape.rows * out_shape.cols * sizeof(float);
    
    // 변경
    if (out_host != nullptr) {
        CUDA_CHECK(hipMemcpy(out_host, tensors[final_output_id], out_bytes, hipMemcpyDeviceToHost));
    }
}