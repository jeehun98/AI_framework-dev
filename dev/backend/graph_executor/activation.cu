#include "hip/hip_runtime.h"
// activation.cu
#include <stdio.h>
#include <math.h>
#include "activation.cuh"

__global__ void activation_relu(const float* input, const float* bias, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        int col = idx % cols;
        float val = input[idx] + (bias ? bias[col] : 0.0f);  // ✅ bias null-safe
        output[idx] = val > 0 ? val : 0;
    }
}

__global__ void activation_sigmoid(const float* input, const float* bias, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        int col = idx % cols;
        float val = input[idx] + (bias ? bias[col] : 0.0f);  // ✅ bias null-safe
        output[idx] = 1.0f / (1.0f + __expf(-val));          // ✅ fast CUDA sigmoid
    }
}

__global__ void activation_tanh(const float* input, const float* bias, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        int col = idx % cols;
        float val = input[idx] + (bias ? bias[col] : 0.0f);  // ✅ bias null-safe
        output[idx] = tanhf(val);
    }
}
