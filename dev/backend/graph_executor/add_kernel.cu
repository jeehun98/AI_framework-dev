#include "hip/hip_runtime.h"
#include "add_kernel.cuh"

/**
 * @brief Element-wise Add with row broadcasting:
 *        output[i][j] = input[i][j] + bias[0][j]
 */
__global__ void add_kernel(const float* input, const float* bias, float* output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = rows * cols;

    if (idx < total) {
        int col = idx % cols;
        output[idx] = input[idx] + bias[col];
    }
}
