#include "hip/hip_runtime.h"
#include "ge_v2_api.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblaslt.h>
#include <mutex>

// -------------------------- 공통 파라미터 블록 --------------------------
struct GemmBiasActParams {
  int M;         // A: MxK, B: KxN, D: MxN
  int N;
  int K;
  int has_bias;  // 0/1
  int act;       // 0:none, 1:ReLU
};

// Row-major 지정 헬퍼
static inline void set_row_major(hipblasLtMatrixLayout_t lay) {
  hipblasLtOrder_t order = HIPBLASLT_ORDER_ROW;
  hipblasLtMatrixLayoutSetAttribute(
      lay, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
}

// -------------------------- f32 스모크용 --------------------------
__global__ void gemm_bias_act_f32_kernel(
    float* __restrict__ D,
    const float* __restrict__ A,
    const float* __restrict__ B,
    const float* __restrict__ bias,  // nullable
    int M, int N, int K,
    int has_bias, int act) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= M || col >= N) return;
  float acc = 0.f;
  for (int k = 0; k < K; ++k) acc = fmaf(A[row*K + k], B[k*N + col], acc);
  if (has_bias && bias) acc += bias[col];
  if (act == 1 && acc < 0.f) acc = 0.f;
  D[row*N + col] = acc;
}

static int launch_gemm_bias_act_f32(
    const ge2_uintptr* bufs, int n, hipStream_t stream) {
  if (n < 4) return -1;
  const auto* p = reinterpret_cast<const GemmBiasActParams*>(bufs[n - 1]);
  if (!p) return -1;

  const float* A = reinterpret_cast<const float*>(bufs[0]); // MxK
  const float* B = reinterpret_cast<const float*>(bufs[1]); // KxN
  const float* bias = nullptr;
  int idxD;
  if (p->has_bias) {
    if (n < 5) return -1;
    bias = reinterpret_cast<const float*>(bufs[2]); // N
    idxD = 3;
  } else {
    idxD = 2;
  }
  float* D = reinterpret_cast<float*>(bufs[idxD]); // MxN

  dim3 blk(16,16), grd((p->N+15)/16, (p->M+15)/16);
  gemm_bias_act_f32_kernel<<<grd, blk, 0, stream>>>(D, A, B, bias,
      p->M, p->N, p->K, p->has_bias, p->act);
  return (hipGetLastError() == hipSuccess) ? 0 : -2;
}

extern "C" int ge2_launch_gemm_bias_act_f32(
    const ge2_uintptr* bufs, int n, void* stream) {
  if (n < 4) return -1;
  return launch_gemm_bias_act_f32(bufs, n, reinterpret_cast<hipStream_t>(stream));
}

// -------------------------- f16 + cuBLASLt ------------------------------

// 후처리 커널들 (폴백용)
__global__ void add_bias_fp16(
    __half* __restrict__ D,
    const float* __restrict__ bias,
    int M, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int size = M * N;
  for (int idx = tid; idx < size; idx += blockDim.x * gridDim.x) {
    int col = idx % N;
    float v = __half2float(D[idx]);
    v += bias[col];
    D[idx] = __float2half(v);
  }
}

__global__ void relu_only_fp16(__half* __restrict__ D, int M, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int size = M * N;
  for (int idx = tid; idx < size; idx += blockDim.x * gridDim.x) {
    float v = __half2float(D[idx]);
    if (v < 0.f) v = 0.f;
    D[idx] = __float2half(v);
  }
}

__global__ void add_bias_relu_fp16(
    __half* __restrict__ D,
    const float* __restrict__ bias,
    int M, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int size = M * N;
  for (int idx = tid; idx < size; idx += blockDim.x * gridDim.x) {
    int col = idx % N;
    float v = __half2float(D[idx]);
    v += bias[col];
    if (v < 0.f) v = 0.f;
    D[idx] = __float2half(v);
  }
}

// 핸들 캐싱
static hipblasLtHandle_t get_lt_handle() {
  static hipblasLtHandle_t handle = nullptr;
  static std::once_flag once;
  std::call_once(once, [](){ hipblasLtCreate(&handle); });
  return handle;
}

extern "C" int ge2_launch_gemm_bias_act_tc_f16(
    const ge2_uintptr* bufs, int n, void* stream_opaque) {
  if (n < 4) return -1;
  hipStream_t stream = reinterpret_cast<hipStream_t>(stream_opaque);

  const auto* p = reinterpret_cast<const GemmBiasActParams*>(bufs[n - 1]);
  if (!p) return -1;

  const __half* A = reinterpret_cast<const __half*>(bufs[0]);
  const __half* B = reinterpret_cast<const __half*>(bufs[1]);
  int idxD = p->has_bias ? 3 : 2;
  __half* D = reinterpret_cast<__half*>(bufs[idxD]);

  const float* bias_f32 = nullptr;
  if (p->has_bias) {
    if (n < 5) return -1;
    bias_f32 = reinterpret_cast<const float*>(bufs[2]);
  }

  const int64_t M = p->M, N = p->N, K = p->K;
  if (M <= 0 || N <= 0 || K <= 0) return -1;

  hipblasLtHandle_t handle = get_lt_handle();
  if (!handle) return -2;

  // 람다: GEMM 실행 (try_epilogue_bias=true이면 Bias/ReluBias 에필로그 시도)
  auto run_matmul = [&](bool try_epilogue_bias, hipblasStatus_t& out_st) -> bool {
    hipblasLtMatmulDesc_t opDesc = nullptr;
    out_st = hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    if (out_st != HIPBLAS_STATUS_SUCCESS) return false;

    // pointer mode host
    hipblasLtPointerMode_t pm = HIPBLASLT_POINTER_MODE_HOST;
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &pm, sizeof(pm));

    hipblasOperation_t transN = HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transN, sizeof(transN));
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transN, sizeof(transN));

    if (try_epilogue_bias && p->has_bias) {
      hipblasLtEpilogue_t epi =
          (p->act == 1) ? HIPBLASLT_EPILOGUE_RELU_BIAS : HIPBLASLT_EPILOGUE_BIAS;
      hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epi, sizeof(epi));
      const void* biasDev = reinterpret_cast<const void*>(bias_f32);
      hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &biasDev, sizeof(biasDev));
    }

    hipblasLtMatrixLayout_t aDesc=nullptr, bDesc=nullptr, cDesc=nullptr, dDesc=nullptr;
    hipblasLtMatrixLayoutCreate(&aDesc, HIP_R_16F, M, K, K);
    hipblasLtMatrixLayoutCreate(&bDesc, HIP_R_16F, K, N, N);
    hipblasLtMatrixLayoutCreate(&cDesc, HIP_R_16F, M, N, N);
    hipblasLtMatrixLayoutCreate(&dDesc, HIP_R_16F, M, N, N);
    set_row_major(aDesc); set_row_major(bDesc); set_row_major(cDesc); set_row_major(dDesc);

    hipblasLtMatmulPreference_t pref = nullptr;
    hipblasLtMatmulPreferenceCreate(&pref);
    size_t max_ws = 0;
    hipblasLtMatmulPreferenceSetAttribute(
        pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &max_ws, sizeof(max_ws));

    hipblasLtMatmulHeuristicResult_t heur[8]; int returned = 0;
    out_st = hipblasLtMatmulAlgoGetHeuristic(
        handle, opDesc, aDesc, bDesc, cDesc, dDesc, pref, 8, heur, &returned);

    void* ws = nullptr;
    if (out_st == HIPBLAS_STATUS_SUCCESS && returned > 0 && heur[0].workspaceSize > 0) {
      hipMalloc(&ws, heur[0].workspaceSize);
    }

    float alpha = 1.0f, beta = 0.0f;
    out_st = hipblasLtMatmul(handle, opDesc,
                            &alpha,
                            A, aDesc,
                            B, bDesc,
                            &beta,
                            D, cDesc,
                            D, dDesc,
                            (returned>0? &heur[0].algo: nullptr),
                            ws, (ws ? heur[0].workspaceSize : 0),
                            stream);

    if (ws) hipFree(ws);
    hipblasLtMatmulPreferenceDestroy(pref);
    if (aDesc) hipblasLtMatrixLayoutDestroy(aDesc);
    if (bDesc) hipblasLtMatrixLayoutDestroy(bDesc);
    if (cDesc) hipblasLtMatrixLayoutDestroy(cDesc);
    if (dDesc) hipblasLtMatrixLayoutDestroy(dDesc);
    hipblasLtMatmulDescDestroy(opDesc);
    return (out_st == HIPBLAS_STATUS_SUCCESS);
  };

  // 1) 에필로그 시도
  hipblasStatus_t st = HIPBLAS_STATUS_SUCCESS;
  bool ok = run_matmul(p->has_bias, st);

  // 2) 실패 시 폴백: 에필로그 OFF + 후처리 커널
  if (!ok) {
    ok = run_matmul(false, st);
    if (!ok) return -2;

    int threads = 256;
    int blocks  = (int)((M * N + threads - 1) / threads);
    if (p->has_bias && p->act == 1) {
      add_bias_relu_fp16<<<blocks, threads, 0, stream>>>(D, bias_f32, (int)M, (int)N);
    } else if (p->has_bias && p->act == 0) {
      add_bias_fp16<<<blocks, threads, 0, stream>>>(D, bias_f32, (int)M, (int)N);
    } else if (!p->has_bias && p->act == 1) {
      relu_only_fp16<<<blocks, threads, 0, stream>>>(D, (int)M, (int)N);
    }
    if (hipGetLastError() != hipSuccess) return -2;
    return 0;
  }

  // 3) 에필로그 성공 + ReLU 단독 케이스
  if (!p->has_bias && p->act == 1) {
    int threads = 256;
    int blocks  = (int)((M * N + threads - 1) / threads);
    relu_only_fp16<<<blocks, threads, 0, stream>>>(D, (int)M, (int)N);
    if (hipGetLastError() != hipSuccess) return -2;
  }

  return 0;
}
