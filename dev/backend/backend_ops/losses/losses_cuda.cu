#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/pytypes.h>
#include <hip/hip_runtime.h>
#include <cmath>

namespace py = pybind11;
#define BLOCK_SIZE 256

float* get_device_ptr(py::object cupy_array) {
    auto interface = cupy_array.attr("__cuda_array_interface__").cast<py::dict>();
    uintptr_t ptr = interface["data"].cast<std::pair<uintptr_t, bool>>().first;
    return reinterpret_cast<float*>(ptr);
}

__global__ void mseLossKernel(const float* y_true, const float* y_pred, float* loss, int n) {
    __shared__ float partial_sum[BLOCK_SIZE];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    float diff = 0.0f;
    float sq = 0.0f;
    if (idx < n) {
        diff = y_pred[idx] - y_true[idx];

        // ✅ NaN 또는 inf 보호
        if (!isnan(diff) && !isinf(diff)) {
            sq = diff * diff;

            // ✅ overflow 방지 (상한 제한)
            if (sq > 1e10f) sq = 1e10f;
        }
        partial_sum[tid] = sq;
    } else {
        partial_sum[tid] = 0.0f;
    }

    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partial_sum[tid] += partial_sum[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) atomicAdd(loss, partial_sum[0]);
}


__global__ void mseGradKernel(const float* y_true, const float* y_pred, float* grad, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) grad[idx] = 2.0f * (y_pred[idx] - y_true[idx]) / n;
}

float mse_loss(py::object y_true, py::object y_pred) {
    float* d_y_true = get_device_ptr(y_true);
    float* d_y_pred = get_device_ptr(y_pred);
    auto shape = y_true.attr("shape").cast<py::tuple>();
    int n = 1;
    for (auto s : shape) n *= s.cast<int>();

    float* d_loss;
    float h_loss = 0.0f;
    hipMalloc(&d_loss, sizeof(float));
    hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice);

    int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    mseLossKernel<<<gridSize, BLOCK_SIZE>>>(d_y_true, d_y_pred, d_loss, n);
    hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_loss);
    return h_loss / n;
}

void mse_grad(py::object y_true, py::object y_pred, py::object grad_out) {
    float* d_y_true = get_device_ptr(y_true);
    float* d_y_pred = get_device_ptr(y_pred);
    float* d_grad = get_device_ptr(grad_out);
    auto shape = y_true.attr("shape").cast<py::tuple>();
    int n = 1;
    for (auto s : shape) n *= s.cast<int>();

    int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    mseGradKernel<<<gridSize, BLOCK_SIZE>>>(d_y_true, d_y_pred, d_grad, n);
}

PYBIND11_MODULE(losses_cuda, m) {
    m.def("mse_loss", &mse_loss, "CuPy 기반 MSE 손실 계산");
    m.def("mse_grad", &mse_grad, "CuPy 기반 MSE 손실 gradient 계산");
}
