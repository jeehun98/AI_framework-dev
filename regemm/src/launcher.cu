#include <hip/hip_runtime.h>
#include "regemm/api.h"

namespace regemm {
void launch_gemm_bias_act_f32_smoke(const GemmBiasActParams&, hipStream_t);

int gemm_bias_act(const GemmBiasActParams& p, void* stream) {
  hipStream_t s = reinterpret_cast<hipStream_t>(stream);
  if (p.dtype == DType::F32) {
    launch_gemm_bias_act_f32_smoke(p, s);
    return hipPeekAtLastError();
  }
  // TODO: F16/BF16/WMMA 경로 선택
  return 1; // unsupported dtype
}
} // namespace regemm
