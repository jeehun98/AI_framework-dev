#include <hip/hip_runtime.h>
#include "../api/epilogue.h"

namespace epi {
Status run_fp32(const Plan& plan, const Tensors& ts, DType bdt, void* stream);
Status run_fp16(const Plan& plan, const Tensors& ts, DType bdt, void* stream);

Status run(const Plan& plan, const Tensors& ts,
           DType xdt, DType ydt, DType bdt, void* stream){
  if (xdt==DType::F32 && ydt==DType::F32) return run_fp32(plan, ts, bdt, stream);
  if (xdt==DType::F16 && ydt==DType::F16) return run_fp16(plan, ts, bdt, stream);
  return {false, "Unsupported dtype combo (MVP supports F32->F32 or F16->F16)"};
}

} // namespace epi
