#include "hip/hip_runtime.h"
// backends/cuda/ops/pool2d/launcher.cu
#include <hip/hip_runtime.h>
#include "backends/cuda/ops/pool2d/api.hpp"

namespace ai {

static inline bool is_nchw_f32_4d_cuda(const Tensor& t){
  return t.device==Device::CUDA && t.desc.dtype==DType::F32 &&
         t.desc.layout==Layout::RowMajor && t.desc.shape.size()==4;
}
static inline bool is_nchw_i32_4d_cuda(const Tensor& t){
  return t.device==Device::CUDA && t.desc.dtype==DType::I32 &&
         t.desc.layout==Layout::RowMajor && t.desc.shape.size()==4;
}

static inline hipStream_t to_cuda(StreamHandle h){ return (hipStream_t)h; }

// ===== 커널 런처 (kernels.cu 와 정확히 동일 시그니처) =====
void maxpool2d_fwd_kernel_launcher(const float*, float*, int32_t*,
                                   int,int,int,int,int,int,int,int,int,int,int,int,bool,
                                   hipStream_t);
void maxpool2d_bwd_kernel_launcher(const float*, const int32_t*, float*,
                                   int,int,int,int,int,int,int,int,int,int,int,int,bool,
                                   hipStream_t);
void avgpool2d_fwd_kernel_launcher(const float*, float*,
                                   int,int,int,int,int,int,int,int,int,int,int,int,bool,bool,
                                   hipStream_t);
void avgpool2d_bwd_kernel_launcher(const float*, float*,
                                   int,int,int,int,int,int,int,int,int,int,int,int,bool,bool,
                                   hipStream_t);

// ===== 출력 크기 공식(커널과 동일) =====
static inline int div_up_host(int a, int b){ return (a + b - 1) / b; }
static inline void out_dims_host(
  int H,int W,int kH,int kW,int sH,int sW,int pH,int pW,int dH,int dW,bool ceil_mode,
  int& Ho,int& Wo)
{
  const int effKH = (kH - 1) * dH + 1;
  const int effKW = (kW - 1) * dW + 1;
  const int aH = H + 2 * pH - effKH;
  const int aW = W + 2 * pW - effKW;
  if (ceil_mode) {
    Ho = (aH >= 0 ? div_up_host(aH, sH) + 1 : 0);
    Wo = (aW >= 0 ? div_up_host(aW, sW) + 1 : 0);
  } else {
    Ho = (aH >= 0 ? (aH / sH) + 1 : 0);
    Wo = (aW >= 0 ? (aW / sW) + 1 : 0);
  }
  if (Ho < 0) Ho = 0;
  if (Wo < 0) Wo = 0;
}

// ================= MaxPool2D =================
Status MaxPool2DCudaLaunch(const Tensor& X, Tensor& Y, Tensor* Indices,
                           const Pool2DAttrs& a, StreamHandle stream)
{
  if (!is_nchw_f32_4d_cuda(X) || !is_nchw_f32_4d_cuda(Y)) return Status::Invalid;
  if (X.desc.shape[0]!=Y.desc.shape[0] || X.desc.shape[1]!=Y.desc.shape[1]) return Status::ShapeMismatch;

  // Y가 기대하는 (Ho,Wo)인지 확인(버그 조기 탐지)
  {
    int Ho=0, Wo=0;
    out_dims_host((int)X.desc.shape[2], (int)X.desc.shape[3],
                  a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode, Ho, Wo);
    if ((int)Y.desc.shape[2]!=Ho || (int)Y.desc.shape[3]!=Wo) return Status::ShapeMismatch;
  }

  int32_t* ind = nullptr;
  if (Indices){
    if (!is_nchw_i32_4d_cuda(*Indices)) return Status::Invalid;
    // Indices 모양은 Y와 동일해야 함
    if (Indices->desc.shape != Y.desc.shape) return Status::ShapeMismatch;
    ind = static_cast<int32_t*>(Indices->data);
  }

  maxpool2d_fwd_kernel_launcher(
    static_cast<const float*>(X.data),
    static_cast<float*>(Y.data),
    ind,
    (int)X.desc.shape[0], (int)X.desc.shape[1], (int)X.desc.shape[2], (int)X.desc.shape[3],
    a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode,
    to_cuda(stream)
  );

  hipError_t e = hipPeekAtLastError();
  if (e != hipSuccess) return Status::RuntimeError;
  return Status::Ok;
}

Status MaxPool2DBackwardCudaLaunch(const Tensor& dY, const Tensor& Indices, Tensor& dX,
                                   const Pool2DAttrs& a, StreamHandle stream)
{
  if (!is_nchw_f32_4d_cuda(dY) || !is_nchw_i32_4d_cuda(Indices) || !is_nchw_f32_4d_cuda(dX))
    return Status::Invalid;

  // dX: (N,C,H,W), dY/Indices: (N,C,Ho,Wo)
  const int N  = (int)dX.desc.shape[0];
  const int C  = (int)dX.desc.shape[1];
  const int H  = (int)dX.desc.shape[2];
  const int W  = (int)dX.desc.shape[3];

  if ((int)dY.desc.shape[0]!=N || (int)dY.desc.shape[1]!=C) return Status::ShapeMismatch;
  if (Indices.desc.shape != dY.desc.shape) return Status::ShapeMismatch;

  // 포워드 공식으로 Ho,Wo 계산하고 dY와 일치 확인
  int Ho=0, Wo=0;
  out_dims_host(H, W, a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode, Ho, Wo);
  if ((int)dY.desc.shape[2]!=Ho || (int)dY.desc.shape[3]!=Wo) return Status::ShapeMismatch;

  // dX = 0 초기화 (atomicAdd 누적)
  hipError_t e = hipMemsetAsync(dX.data, 0,
    sizeof(float) * (size_t)N*C*H*W, to_cuda(stream));
  if (e != hipSuccess) return Status::Invalid;

  maxpool2d_bwd_kernel_launcher(
    static_cast<const float*>(dY.data),
    static_cast<const int32_t*>(Indices.data),
    static_cast<float*>(dX.data),
    N, C, H, W,
    a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode,
    to_cuda(stream)
  );

  e = hipPeekAtLastError();
  if (e != hipSuccess) return Status::RuntimeError;
  return Status::Ok;
}

// ================= AvgPool2D =================
Status AvgPool2DCudaLaunch(const Tensor& X, Tensor& Y,
                           const Pool2DAttrs& a, StreamHandle stream)
{
  if (!is_nchw_f32_4d_cuda(X) || !is_nchw_f32_4d_cuda(Y)) return Status::Invalid;
  if (X.desc.shape[0]!=Y.desc.shape[0] || X.desc.shape[1]!=Y.desc.shape[1]) return Status::ShapeMismatch;

  // Y 모양 검증
  {
    int Ho=0, Wo=0;
    out_dims_host((int)X.desc.shape[2], (int)X.desc.shape[3],
                  a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode, Ho, Wo);
    if ((int)Y.desc.shape[2]!=Ho || (int)Y.desc.shape[3]!=Wo) return Status::ShapeMismatch;
  }

  avgpool2d_fwd_kernel_launcher(
    static_cast<const float*>(X.data),
    static_cast<float*>(Y.data),
    (int)X.desc.shape[0], (int)X.desc.shape[1], (int)X.desc.shape[2], (int)X.desc.shape[3],
    a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode, a.count_include_pad,
    to_cuda(stream)
  );
  hipError_t e = hipPeekAtLastError();
  if (e != hipSuccess) return Status::RuntimeError;
  return Status::Ok;
}

Status AvgPool2DBackwardCudaLaunch(const Tensor& dY, Tensor& dX,
                                   const Pool2DAttrs& a, StreamHandle stream)
{
  if (!is_nchw_f32_4d_cuda(dY) || !is_nchw_f32_4d_cuda(dX)) return Status::Invalid;

  // (N,C)만 일치하면 됨. 공간축은 dX(H,W), dY(Ho,Wo)
  if ((int)dY.desc.shape[0]!=(int)dX.desc.shape[0] ||
      (int)dY.desc.shape[1]!=(int)dX.desc.shape[1]) return Status::ShapeMismatch;

  // 기대 dY(Ho,Wo) 검증
  {
    int Ho=0, Wo=0;
    out_dims_host((int)dX.desc.shape[2], (int)dX.desc.shape[3],
                  a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode, Ho, Wo);
    if ((int)dY.desc.shape[2]!=Ho || (int)dY.desc.shape[3]!=Wo) return Status::ShapeMismatch;
  }

  // dX = 0 초기화 (atomicAdd)
  hipError_t e = hipMemsetAsync(dX.data, 0,
    sizeof(float) * (size_t)dX.desc.shape[0]*dX.desc.shape[1]*dX.desc.shape[2]*dX.desc.shape[3],
    to_cuda(stream));
  if (e != hipSuccess) return Status::Invalid;

  avgpool2d_bwd_kernel_launcher(
    static_cast<const float*>(dY.data),
    static_cast<float*>(dX.data),
    (int)dX.desc.shape[0], (int)dX.desc.shape[1], (int)dX.desc.shape[2], (int)dX.desc.shape[3],
    a.kH,a.kW,a.sH,a.sW,a.pH,a.pW,a.dH,a.dW,a.ceil_mode, a.count_include_pad,
    to_cuda(stream)
  );
  e = hipPeekAtLastError();
  if (e != hipSuccess) return Status::RuntimeError;
  return Status::Ok;
}

} // namespace ai
