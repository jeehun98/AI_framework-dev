#include "hip/hip_runtime.h"
// backends/cuda/ops/gemm/launcher.cu
// (FWD+BWD 통합, workspace 지원 / 정책화 디스패치 + BiasMode 런타임→컴파일타임 브릿지)
#include <hip/hip_runtime.h>
#include <cstring>
#include <stdexcept>
#include <limits>

// NVTX 공용 shim
#include "backends/cuda/ops/_common/shim/nvtx.hpp"
#include "backends/cuda/ops/gemm/detail/nvtx_shim.h" // ← 추가 (NVTX_COLOR, NVTX_MARK 제공)

#include "backends/cuda/ops/_common/shim/ai_shim.hpp"
#include "backends/cuda/ops/gemm/detail/gemm_common.hpp"
#include "backends/cuda/ops/gemm/detail/config.h"     // REGEMM_* 타일/블록 매크로
#include "backends/cuda/ops/gemm/detail/api.h"
#include "backends/cuda/ops/gemm/detail/traits.hpp"   // BiasMode / to_bias_mode / Epilogue 정책
#include "backends/cuda/ops/gemm/api.hpp"             // GemmWorkspace, GemmCudaLaunch/Backward
#include "backends/cuda/ops/epilogue/api.hpp"         // ⟵ 추가: Standalone Epilogue
 
#ifndef AI_RETURN_IF_ERROR
#define AI_RETURN_IF_ERROR(expr)                          \
  do {                                                    \
    ::ai::Status _st__ = (expr);                          \
    if (_st__ != ::ai::Status::Ok) return _st__;          \
  } while (0)
#endif


// 폴백 사용 플래그(정책): 필요 시 외부에서 오버라이드
#ifndef REGEMM_USE_STANDALONE_EPILOGUE_FALLBACK
#define REGEMM_USE_STANDALONE_EPILOGUE_FALLBACK 1
#endif

//
// 커널 선언(템플릿 인스턴스 디스패치용) — 정의는 kernels/*.cu에 존재
//
namespace regemm {
  // Non-EX (C 사용 여부/SaveZ 없음)
  template<int BM_, int BN_, int BK_, ActKind AK, BiasMode BM, bool HasC>
  __global__ void gemm_bias_act_f32_tiled_kernel(GemmBiasActParams p);
  void launch_gemm_bias_act_f32_smoke (const GemmBiasActParams& p, hipStream_t s);

  // EX (Z stash 포함)
  template<int BM_, int BN_, int BK_, ActKind AK, BiasMode BM, bool HasC, bool SaveZ>
  __global__ void gemm_bias_act_f32_tiled_kernel_ex(GemmBiasActParamsEx p);
  void launch_gemm_bias_act_f32_smoke_ex (const GemmBiasActParamsEx& p, hipStream_t s);

  // BWD (정의는 kernels/regemm_backward.cu)
  void gemm_bias_act_bwd_f32(const GemmBiasActBwdParams& p, hipStream_t s);
} // namespace regemm

namespace {
using namespace ai::gemm_common;

// --- 관대 추론(1D/2D, (1,N),(M,1),(1,1) 허용) ---
inline regemm::BiasKind infer_bias_kind_fallback(const ai::Tensor* Bias, int64_t M, int64_t N) {
  using BK = regemm::BiasKind;
  if (!Bias || !Bias->data) return BK::None;
  const auto& s = Bias->desc.shape;
  int64_t numel = 1;
  for (auto v : s) numel *= v;
  if (numel <= 0) return BK::None;

  // 정확 매칭
  if (s.size()==2 && s[0]==1 && s[1]==N) return BK::PerN;
  if (s.size()==1 && s[0]==N)            return BK::PerN;
  if (s.size()==2 && s[0]==M && s[1]==1) return BK::PerM;
  if (s.size()==1 && s[0]==M)            return BK::PerM;
  if ((s.size()==2 && s[0]==1 && s[1]==1) ||
      (s.size()==1 && s[0]==1))          return BK::Scalar;

  // 느슨한 보정: numel 기준
  if (numel == N) return BK::PerN;
  if (numel == M) return BK::PerM;
  if (numel == 1) return BK::Scalar;

  return BK::None;
}

// Bias kind 최종 결정: 엄격(1D lenMN) → 실패 시 관대
inline regemm::BiasKind decide_bias_kind(const ai::Tensor* Bias, int64_t M, int64_t N) {
  auto strict = infer_bias_kind_1d_lenMN(Bias, M, N);
  if (strict != regemm::BiasKind::None) return strict;
  return infer_bias_kind_fallback(Bias, M, N);
}

// Bias 버퍼 크기/형식 검증(가능하면)
inline bool validate_bias_buffer(const ai::Tensor* Bias, int64_t M, int64_t N,
                                 regemm::BiasKind kind) {
  if (!Bias || !Bias->data || kind == regemm::BiasKind::None) return true;
  if (Bias->desc.dtype != ai::DType::F32) return false;

  const size_t need = regemm::expected_bias_elems(static_cast<int>(M), static_cast<int>(N), kind);
  if (need == 0) return false;

  size_t numel = 1;
  for (auto v : Bias->desc.shape) numel *= static_cast<size_t>(v);
  if (numel != need) {
    // (1,N)/(M,1)/(1,1) 관대 허용
    if (!(Bias->desc.shape.size()==2 &&
          ((kind==regemm::BiasKind::PerN && Bias->desc.shape[0]==1 && Bias->desc.shape[1]==N) ||
           (kind==regemm::BiasKind::PerM && Bias->desc.shape[0]==M && Bias->desc.shape[1]==1) ||
           (kind==regemm::BiasKind::Scalar && Bias->desc.shape[0]==1 && Bias->desc.shape[1]==1)))) {
      return false;
    }
  }
  return true;
}

inline bool validate_ws_lt(const ai::GemmWorkspace* ws) {
  if (!ws) return true;
  if (ws->lt_workspace && !regemm::is_workspace_aligned(ws->lt_workspace, 256)) return false;
  return true;
}

inline bool validate_ws_scratch(const ai::GemmWorkspace* ws, int64_t M, int64_t N) {
  if (!ws || !ws->scratch) return true;
  if (ws->scratch_bytes > 0) {
    const size_t need = static_cast<size_t>(M) * static_cast<size_t>(N) * sizeof(float);
    if (ws->scratch_bytes < need) return false;
  }
  return true;
}

// 타일/블록 파라미터(런처에서도 사용)
constexpr int BM  = REGEMM_TILE_M;
constexpr int BN  = REGEMM_TILE_N;
constexpr int BK  = REGEMM_TILE_K;
constexpr int TDX = REGEMM_BLOCK_TDX;
constexpr int TDY = REGEMM_BLOCK_TDY;

// === [EX 디스패치 헬퍼들] ===
// 고정된 BiasMode 인자로 직접 커널 호출
template<regemm::ActKind AK, regemm::BiasMode BMmode, bool HasC, bool SaveZ>
inline void launch_ex_cfg(const regemm::GemmBiasActParamsEx& p, hipStream_t s) {
  dim3 block(TDX, TDY);
  dim3 grid((p.N + BN - 1) / BN, (p.M + BM - 1) / BM);
  regemm::gemm_bias_act_f32_tiled_kernel_ex<BM, BN, BK, AK, BMmode, HasC, SaveZ><<<grid, block, 0, s>>>(p);
}

// 런타임 BiasMode → 컴파일타임 인스턴스 분배
template<regemm::ActKind AK, bool SaveZ>
inline void launch_ex_cfg_bm(const regemm::GemmBiasActParamsEx& p,
                             regemm::BiasMode bm,
                             hipStream_t s) {
  constexpr bool HasC = false; // FWD에서는 C 미사용
  switch (bm) {
    case regemm::BiasMode::PerM:
      launch_ex_cfg<AK, regemm::BiasMode::PerM, HasC, SaveZ>(p, s); break;
    case regemm::BiasMode::PerN:
      launch_ex_cfg<AK, regemm::BiasMode::PerN, HasC, SaveZ>(p, s); break;
    case regemm::BiasMode::Full: // (Scalar)
      launch_ex_cfg<AK, regemm::BiasMode::Full, HasC, SaveZ>(p, s); break;
    case regemm::BiasMode::None:
    default:
      launch_ex_cfg<AK, regemm::BiasMode::None, HasC, SaveZ>(p, s); break;
  }
}

// (참고) Non-EX FWD 디스패치(현재는 사용하지 않음 — EX로 통합)
template<regemm::ActKind AK, regemm::BiasMode BMmode, bool HasC>
inline void launch_fwd_cfg(const regemm::GemmBiasActParams& p, hipStream_t s) {
  dim3 block(TDX, TDY);
  dim3 grid((p.N + BN - 1) / BN, (p.M + BM - 1) / BM);
  regemm::gemm_bias_act_f32_tiled_kernel<BM, BN, BK, AK, BMmode, HasC><<<grid, block, 0, s>>>(p);
}

} // anonymous

namespace ai {

// =========================
// Forward (save_z + Lt WS 지원 / 정책화 디스패치)
// =========================
ai::Status GemmCudaLaunch(
    const Tensor& A, const Tensor& B, const Tensor* Bias /*=nullptr*/,
    Tensor& Y, const GemmAttrs& attrs,
    StreamHandle stream,
    Tensor* Z_saved /*=nullptr*/,
    const GemmWorkspace* ws /*=nullptr*/
) {
  NVTX_RANGE("gemm.fwd", NVTX_COLOR::Orange);

  // 1) 디바이스/형식/레이아웃 체크
  if (!is_cuda_f32_rowmajor(A) || !is_cuda_f32_rowmajor(B) || !is_cuda_f32_rowmajor(Y))
    return ai::Status::DeviceMismatch;
  if (attrs.trans_a || attrs.trans_b) return ai::Status::TransposeNotSupported;

  // 2) shape
  if (A.desc.shape.size()!=2 || B.desc.shape.size()!=2 || Y.desc.shape.size()!=2)
    return ai::Status::ShapeMismatch;
  const int64_t M  = A.desc.shape[0];
  const int64_t K  = A.desc.shape[1];
  const int64_t Kb = B.desc.shape[0];
  const int64_t N  = B.desc.shape[1];
  if (K!=Kb || Y.desc.shape[0]!=M || Y.desc.shape[1]!=N) return ai::Status::ShapeMismatch;

  // 3) leading dims
  const int64_t lda = infer_ld_rowmajor_2d(A);
  const int64_t ldb = infer_ld_rowmajor_2d(B);
  const int64_t ldd = infer_ld_rowmajor_2d(Y);
  if (lda < K || ldb < N || ldd < N) return ai::Status::StrideMismatch;

  // 4) int32 범위
  if (!fits_int32(M) || !fits_int32(N) || !fits_int32(K) ||
      !fits_int32(lda) || !fits_int32(ldb) || !fits_int32(ldd)) {
    return ai::Status::Invalid;
  }

  // 5) Z 저장 여부 및 검증
  if (attrs.save_z && Z_saved == nullptr) return ai::Status::MissingOutput;
  const bool want_save_z = attrs.save_z && (Z_saved != nullptr);

  int   ldZ_i = 0;
  void* Z_ptr = nullptr;
  if (want_save_z) {
    if (!is_cuda_f32_rowmajor(*Z_saved)) return ai::Status::DeviceMismatch;
    if (Z_saved->desc.shape.size()!=2 ||
        Z_saved->desc.shape[0]!=M || Z_saved->desc.shape[1]!=N) {
      return ai::Status::ShapeMismatch;
    }
    const bool alias_Y = (Z_saved->data == Y.data);
    const int64_t ldZ = alias_Y ? ldd : infer_ld_rowmajor_2d(*Z_saved);
    if (ldZ < N) return ai::Status::StrideMismatch;
    if (!fits_int32(ldZ)) return ai::Status::Invalid;
    ldZ_i = static_cast<int>(ldZ);
    Z_ptr = Z_saved->data;
  }

  // 5.5) Workspace 가드(정렬/크기)
  if (!validate_ws_lt(ws)) return ai::Status::Invalid;

  // 6) regemm 파라미터 (Ex 경로 사용)
  regemm::GemmBiasActParamsEx p{};
  p.M = static_cast<int>(M);
  p.N = static_cast<int>(N);
  p.K = static_cast<int>(K);

  p.A   = A.data; p.lda = static_cast<int>(lda);
  p.B   = B.data; p.ldb = static_cast<int>(ldb);
  p.C   = nullptr; p.ldc = 0;                // C는 사용 안 함
  p.D   = Y.data; p.ldd = static_cast<int>(ldd);

  p.alpha = 1.0f;
  p.beta  = 0.0f;

  // ---- bias 전달 + kind 추론(엄격 → 관대) ----
  p.bias      = (Bias && Bias->data) ? Bias->data : nullptr;
  p.bias_kind = decide_bias_kind(Bias, M, N);
  if (!validate_bias_buffer(Bias, M, N, p.bias_kind)) return ai::Status::Invalid;

  // ---- activation / leaky slope ----
  p.act         = to_regemm_act(attrs.act);
  p.leaky_slope = attrs.leaky_slope;

  // ---- Z 저장: pre-activation을 단일 패스로 저장 ----
  p.Z           = want_save_z ? Z_ptr : nullptr;
  p.ldZ         = want_save_z ? ldZ_i : 0;   // 0이면 내부에서 ldd로 간주
  p.save_preact = want_save_z ? 1      : 0;

  // ---- Lt workspace (있으면 전달) ----
  p.lt_workspace       = ws ? ws->lt_workspace       : nullptr;
  p.lt_workspace_bytes = ws ? ws->lt_workspace_bytes : 0;

  // 7) 디스패치
  const bool tiny = (p.M * p.N < 4096) || (p.K < 8);
  const hipStream_t cs = reinterpret_cast<hipStream_t>(stream);

  // ===== 7.A 성능 우선: 기존 fused(EX) 경로 =====
  if (!REGEMM_USE_STANDALONE_EPILOGUE_FALLBACK) {
    NVTX_RANGE("gemm.fwd.ex_dispatch", NVTX_COLOR::Teal);
    if (tiny) { regemm::launch_gemm_bias_act_f32_smoke_ex(p, cs); return ai::Status::Ok; }

    const regemm::BiasMode bm = regemm::to_bias_mode(p.bias_kind);
    const bool SaveZ = want_save_z;

    switch (p.act) {
      case regemm::ActKind::ReLU:
        NVTX_MARK("ex.relu");

        if (SaveZ) launch_ex_cfg_bm<regemm::ActKind::ReLU,      true >(p, bm, cs);
        else       launch_ex_cfg_bm<regemm::ActKind::ReLU,      false>(p, bm, cs);
        break;
      case regemm::ActKind::LeakyReLU:
        NVTX_MARK("ex.leakyrelu");

        if (SaveZ) launch_ex_cfg_bm<regemm::ActKind::LeakyReLU, true >(p, bm, cs);
        else       launch_ex_cfg_bm<regemm::ActKind::LeakyReLU, false>(p, bm, cs);
        break;
      case regemm::ActKind::GELU:
        NVTX_MARK("ex.gelu");

        if (SaveZ) launch_ex_cfg_bm<regemm::ActKind::GELU,      true >(p, bm, cs);
        else       launch_ex_cfg_bm<regemm::ActKind::GELU,      false>(p, bm, cs);
        break;
      case regemm::ActKind::Sigmoid:
        NVTX_MARK("ex.sigmoid");

        if (SaveZ) launch_ex_cfg_bm<regemm::ActKind::Sigmoid,   true >(p, bm, cs);
        else       launch_ex_cfg_bm<regemm::ActKind::Sigmoid,   false>(p, bm, cs);
        break;
      case regemm::ActKind::Tanh:
        NVTX_MARK("ex.tanh");

        if (SaveZ) launch_ex_cfg_bm<regemm::ActKind::Tanh,      true >(p, bm, cs);
        else       launch_ex_cfg_bm<regemm::ActKind::Tanh,      false>(p, bm, cs);
        break;
      case regemm::ActKind::None:
      default:
        NVTX_MARK("ex.none");

        if (SaveZ) launch_ex_cfg_bm<regemm::ActKind::None,      true >(p, bm, cs);
        else       launch_ex_cfg_bm<regemm::ActKind::None,      false>(p, bm, cs);
        break;
    }
    return ai::Status::Ok;
  }

  // ===== 7.B 폴백: GEMM(=X) → Epilogue 호출 =====
  NVTX_RANGE("gemm.fwd.fallback", NVTX_COLOR::Gray);

  // (i) pre-activation만 먼저 만든다: EX 커널을 "act=None, bias=None"로 호출하여
  //     X = A*B 를 Y 또는 scratch에 쓴다. (SaveZ=false)
  void* Xbuf = Y.data;
  int   ldX  = static_cast<int>(ldd);
  bool  use_scratch = false;
  if (ws && ws->scratch && ws->scratch_bytes >= size_t(M)*size_t(N)*sizeof(float)) {
    Xbuf = ws->scratch;
    ldX  = static_cast<int>(N); // scratch는 [M,N] 연속 가정
    use_scratch = true;
  }

  // EX 파라미터를 복사해서, bias/act을 제거하고 출력 대상만 Xbuf로 바꾼다.
  regemm::GemmBiasActParamsEx p_x = p;
  p_x.D        = Xbuf;
  p_x.ldd      = ldX;
  p_x.bias     = nullptr;
  p_x.bias_kind= regemm::BiasKind::None;
  p_x.act      = regemm::ActKind::None;
  p_x.save_preact = 0;
  p_x.Z        = nullptr;
  p_x.ldZ      = 0;

  if (tiny) { 
    NVTX_MARK("fallback.tiny_gemm");

    regemm::launch_gemm_bias_act_f32_smoke_ex(p_x, cs); 
  } else {
    NVTX_MARK("fallback.main_gemm");

    launch_ex_cfg_bm<regemm::ActKind::None, false /*SaveZ*/>(p_x, regemm::BiasMode::None, cs);
  }

  // (ii) Standalone Epilogue 실행: Xbuf + Bias + Act (+Z) → Y
  {
    NVTX_RANGE("fallback.epilogue", NVTX_COLOR::Cyan);

    ai::EpilogueAttrs eattr;
    eattr.act         = attrs.act;
    eattr.leaky_slope = attrs.leaky_slope;
    eattr.save_z      = attrs.save_z;

    // regemm::BiasKind → ai::BiasLayout 매핑
    ai::BiasLayout bl = ai::BiasLayout::None;
    switch (p.bias_kind) {
      case regemm::BiasKind::PerM:   bl = ai::BiasLayout::PerM;   break;
      case regemm::BiasKind::PerN:   bl = ai::BiasLayout::PerN;   break;
      case regemm::BiasKind::Scalar: bl = ai::BiasLayout::Scalar; break;
      default:                       bl = ai::BiasLayout::None;   break;
    }

    // EpilogueFwdParams 채우기 (새 API는 raw 포인터/ld 기반)
    const float* bias_ptr = (p.bias && p.bias_kind != regemm::BiasKind::None)
                            ? reinterpret_cast<const float*>(p.bias) : nullptr;

    ai::EpilogueFwdParams ep{};
    ep.X          = reinterpret_cast<const float*>(Xbuf);
    ep.ldX        = ldX;
    ep.Bias       = bias_ptr;
    ep.bias_layout= bl;
    ep.Y          = reinterpret_cast<float*>(Y.data);
    ep.ldY        = static_cast<int>(ldd);
    ep.Z          = want_save_z ? reinterpret_cast<float*>(Z_ptr) : nullptr;
    ep.ldZ        = want_save_z ? ldZ_i : 0;
    ep.M          = static_cast<int>(M);
    ep.N          = static_cast<int>(N);

    AI_RETURN_IF_ERROR( ai::EpilogueFwdLaunch(ep, eattr, stream) );
  }

  // scratch를 썼으면 별도 해제는 상위 WS 정책(캡처-세이프)에서 관리
  return ai::Status::Ok;
}

// =========================
// Backward (원문 경로 유지) — 검증·성능 회귀 방지
// =========================
ai::Status GemmCudaBackward(
    const Tensor& A, const Tensor& B, const Tensor* C,
    const Tensor& gY, const Tensor& Z,
    Tensor* gA, Tensor* gB, Tensor* gC, Tensor* gBias,
    const GemmAttrs& attrs,
    StreamHandle stream,
    const GemmWorkspace* ws /*=nullptr*/
) {
  NVTX_RANGE("gemm.bwd", NVTX_COLOR::Red);

  // 1) 디바이스/타입/레이아웃/transpose
  if (!is_cuda_f32_rowmajor(A) || !is_cuda_f32_rowmajor(B) ||
      !is_cuda_f32_rowmajor(gY) || !is_cuda_f32_rowmajor(Z))
    return ai::Status::DeviceMismatch;
  if (gA && !is_cuda_f32_rowmajor(*gA)) return ai::Status::DeviceMismatch;
  if (gB && !is_cuda_f32_rowmajor(*gB)) return ai::Status::DeviceMismatch;
  if (gC && !is_cuda_f32_rowmajor(*gC)) return ai::Status::DeviceMismatch;
  if (C  && !is_cuda_f32_rowmajor(*C))  return ai::Status::DeviceMismatch;
  if (attrs.trans_a || attrs.trans_b)   return ai::Status::TransposeNotSupported;

  // 2) shape
  if (A.desc.shape.size()!=2 || B.desc.shape.size()!=2 ||
      gY.desc.shape.size()!=2 || Z.desc.shape.size()!=2)
    return ai::Status::ShapeMismatch;

  const int64_t M  = A.desc.shape[0];
  const int64_t K  = A.desc.shape[1];
  const int64_t Kb = B.desc.shape[0];
  const int64_t N  = B.desc.shape[1];
  if (K != Kb) return ai::Status::ShapeMismatch;

  if (gY.desc.shape[0]!=M || gY.desc.shape[1]!=N) return ai::Status::ShapeMismatch;
  if (Z .desc.shape[0]!=M || Z .desc.shape[1]!=N) return ai::Status::ShapeMismatch;

  if (gA && (gA->desc.shape.size()!=2 || gA->desc.shape[0]!=M || gA->desc.shape[1]!=K)) return ai::Status::ShapeMismatch;
  if (gB && (gB->desc.shape.size()!=2 || gB->desc.shape[0]!=K || gB->desc.shape[1]!=N)) return ai::Status::ShapeMismatch;
  if (gC) {
    if (!C) return ai::Status::MissingInput;
    if (gC->desc.shape.size()!=2 || gC->desc.shape[0]!=M || gC->desc.shape[1]!=N) return ai::Status::ShapeMismatch;
  }

  // 3) leading dims
  const int64_t lda  = infer_ld_rowmajor_2d(A);
  const int64_t ldb  = infer_ld_rowmajor_2d(B);
  const int64_t ldgY = infer_ld_rowmajor_2d(gY);
  const int64_t ldZ  = infer_ld_rowmajor_2d(Z);
  if (lda < K || ldb < N || ldgY < N || ldZ < N) return ai::Status::StrideMismatch;

  int64_t ldgA = 0, ldgB = 0, ldgC = 0;
  if (gA) { ldgA = infer_ld_rowmajor_2d(*gA); if (ldgA < K) return ai::Status::StrideMismatch; }
  if (gB) { ldgB = infer_ld_rowmajor_2d(*gB); if (ldgB < N) return ai::Status::StrideMismatch; }
  if (gC) { ldgC = infer_ld_rowmajor_2d(*gC); if (ldgC < N) return ai::Status::StrideMismatch; }

  // int32 범위
  if (!fits_int32(M) || !fits_int32(N) || !fits_int32(K) ||
      !fits_int32(lda) || !fits_int32(ldb) || !fits_int32(ldgY) || !fits_int32(ldZ) ||
      (gA && !fits_int32(ldgA)) || (gB && !fits_int32(ldgB)) || (gC && !fits_int32(ldgC))) {
    return ai::Status::Invalid;
  }

  // 4) gBias kind (gBias 존재 시에만 의미)
  regemm::BiasKind bk = regemm::BiasKind::None;
  if (gBias && gBias->data) {
    bk = decide_bias_kind(gBias, M, N);
    if (!validate_bias_buffer(gBias, M, N, bk)) return ai::Status::Invalid;
  }

  // 4.5) 캡처-세이프 dZ scratch / Lt workspace 검증
  if (!validate_ws_lt(ws)) return ai::Status::Invalid;
  if (!validate_ws_scratch(ws, M, N)) return ai::Status::Invalid;

  float* dZ = nullptr;
  if (ws && ws->scratch) {
    dZ = reinterpret_cast<float*>(ws->scratch);
  }

  // 5) 파라미터
  regemm::GemmBiasActBwdParams p{};
  p.M = static_cast<int>(M);
  p.N = static_cast<int>(N);
  p.K = static_cast<int>(K);

  p.A   = A.data;  p.lda  = static_cast<int>(lda);
  p.B   = B.data;  p.ldb  = static_cast<int>(ldb);
  p.C   = C ? C->data : nullptr;
  p.ldc = C ? static_cast<int>(infer_ld_rowmajor_2d(*C)) : 0;

  p.gY  = gY.data; p.ldgY = static_cast<int>(ldgY);
  p.Z   = Z.data;  p.ldZ  = static_cast<int>(ldZ);

  p.gA  = gA ? gA->data : nullptr;  p.ldgA = gA ? static_cast<int>(ldgA) : 0;
  p.gB  = gB ? gB->data : nullptr;  p.ldgB = gB ? static_cast<int>(ldgB) : 0;
  p.gC  = gC ? gC->data : nullptr;  p.ldgC = gC ? static_cast<int>(ldgC) : 0;
  p.gBias = gBias ? gBias->data : nullptr;

  // 6) 스케일/에필로그
  p.alpha = 1.0f;
  p.beta  = (C && gC) ? 1.0f : 0.0f;

  p.bias_kind   = bk;
  p.act         = to_regemm_act(attrs.act);
  p.leaky_slope = attrs.leaky_slope;

  // 6.5) dZ scratch + Lt WS 전달
  p.gZ_scratch         = dZ;                                  // 외부 제공 시 malloc-free 없음
  p.ldgZ               = (dZ ? static_cast<int>(N) : 0);      // 제공 시 반드시 N
  p.lt_workspace       = ws ? ws->lt_workspace       : nullptr;
  p.lt_workspace_bytes = ws ? ws->lt_workspace_bytes : 0;

  // 7) 실행 (원문 경로 유지)
  {
    NVTX_RANGE("bwd.core", NVTX_COLOR::Magenta);
    regemm::gemm_bias_act_bwd_f32(p, reinterpret_cast<hipStream_t>(stream));
  }
  

  return ai::Status::Ok;
}

} // namespace ai
