#include "hip/hip_runtime.h"
// backends/cuda/ops/sdpa/launcher.cu
#include <hip/hip_runtime.h>
#include <cmath>
#include "backends/cuda/ops/sdpa/api.hpp"
#include "backends/cuda/ops/dropout/api.hpp"
#include "backends/cuda/ops/softmax/api.hpp"
#include "ai/op_schema.hpp"
#include "ai/dispatch.hpp"
#include <cstdint>

namespace ai { namespace ops {
  // GEMM
  int gemm_run(const Tensor& A, const Tensor& B, const Tensor* Bias,
               Tensor& Y, const GemmAttrs& attrs, StreamHandle stream);

  // Softmax (attrs 기반)
  int softmax_run(const Tensor& X, const Tensor* mask, Tensor& Y,
                  const ai::SoftmaxAttrs& attrs, StreamHandle stream);
  int softmax_backward_run(const Tensor& Y, const Tensor& dY, Tensor& dX,
                           const ai::SoftmaxAttrs& attrs, StreamHandle stream);

  // Dropout
  int dropout_run(const Tensor& X, Tensor& Y, Tensor* mask,
                  const ai::DropoutAttrs& attrs, StreamHandle stream);
}}

namespace ai {

static inline bool is_bhxd_f32_4d_cuda(const Tensor& t){
  return t.device==Device::CUDA && t.desc.dtype==DType::F32 &&
         t.desc.layout==Layout::RowMajor && t.desc.shape.size()==4;
}
static inline hipStream_t to_cuda(StreamHandle h){ return reinterpret_cast<hipStream_t>(h); }

// RowMajor 2D transpose: in[R,C] -> out[C,R]
__global__ void transpose_rm_f32(const float* __restrict__ in, float* __restrict__ out,
                                 int R, int C){
  int r = blockIdx.y * blockDim.y + threadIdx.y;
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (r < R && c < C) out[c * R + r] = in[r * C + c];
}

// ---- 로컬 유틸: 4D F32 CUDA 텐서 확인 (RowMajor) ----
static inline bool is4d_f32_cuda(const ai::Tensor& t){
  return t.device==ai::Device::CUDA &&
         t.desc.dtype==ai::DType::F32 &&
         t.desc.layout==ai::Layout::RowMajor &&
         t.desc.shape.size()==4;
}

// causal mask: S[b,h,m,n] += huge_neg if n>m
__global__ void causal_mask_add_kernel(float* S, int B,int H,int M,int N, float huge_neg){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*M*N;
  if (idx >= total) return;
  int t = idx;
  int n = t % N; t /= N;
  int m = t % M; t /= M;
  if (n > m) S[idx] += huge_neg;
}

// zero gS on upper triangle
__global__ void causal_gs_zero_kernel(float* gS, int B,int H,int M,int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*M*N;
  if (idx >= total) return;
  int t = idx;
  int n = t % N; t /= N;
  int m = t % M; t /= M;
  if (n > m) gS[idx] = 0.f;
}

__global__ void scale_kernel(float* X, int64_t n, float s){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) X[i] *= s;
}

// mask: I8/I32/F32 지원. 
__global__ void add_mask_i8_kernel(float* S, const int8_t* M, int B,int H,int Mlen,int N, float huge_neg){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*Mlen*N;
  if (idx >= total) return;
  if (M[idx]) S[idx] += huge_neg;
}
__global__ void add_mask_i32_kernel(float* S, const int32_t* M, int B,int H,int Mlen,int N, float huge_neg){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*Mlen*N;
  if (idx >= total) return;
  if (M[idx]) S[idx] += huge_neg;
}
__global__ void add_mask_f32_kernel(float* S, const float* M, int B,int H,int Mlen,int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*Mlen*N;
  if (idx >= total) return;
  S[idx] += M[idx];
}

// backward에서 gS를 마스크 위치에 0으로
__global__ void zero_gs_mask_i8_kernel(float* gS, const int8_t* M, int B,int H,int Mlen,int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*Mlen*N;
  if (idx >= total) return;
  if (M[idx]) gS[idx] = 0.f;
}
__global__ void zero_gs_mask_i32_kernel(float* gS, const int32_t* M, int B,int H,int Mlen,int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*Mlen*N;
  if (idx >= total) return;
  if (M[idx]) gS[idx] = 0.f;
}
__global__ void zero_gs_mask_f32_kernel(float* gS, const float* M, int B,int H,int Mlen,int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = B*H*Mlen*N;
  if (idx >= total) return;
  if (M[idx] != 0.0f) gS[idx] = 0.f;
}

// =============== Forward ===============
Status SDPACudaLaunch(const Tensor& Q, const Tensor& K, const Tensor& V,
                      const Tensor* mask, Tensor& Y,
                      const SDPAAttrs& attrs, StreamHandle stream)
{
  if (!is_bhxd_f32_4d_cuda(Q) || !is_bhxd_f32_4d_cuda(K) ||
      !is_bhxd_f32_4d_cuda(V) || !is_bhxd_f32_4d_cuda(Y))
    return Status::Invalid;

  const int B  = (int)Q.desc.shape[0];
  const int H  = (int)Q.desc.shape[1];
  const int M  = (int)Q.desc.shape[2];
  const int D  = (int)Q.desc.shape[3];
  const int N  = (int)K.desc.shape[2];

  if (K.desc.shape[0]!=B || K.desc.shape[1]!=H || K.desc.shape[3]!=D) return Status::ShapeMismatch;
  if (V.desc.shape[0]!=B || V.desc.shape[1]!=H || V.desc.shape[2]!=N || V.desc.shape[3]!=D) return Status::ShapeMismatch;
  if (Y.desc.shape[0]!=B || Y.desc.shape[1]!=H || Y.desc.shape[2]!=M  || Y.desc.shape[3]!=D) return Status::ShapeMismatch;

  // workspace
  size_t nScores = (size_t)B*H*M*N;
  float *dS=nullptr, *dP=nullptr, *dKt=nullptr;
  if (hipMalloc(&dS,  sizeof(float)*nScores)!=hipSuccess) return Status::RuntimeError;
  if (hipMalloc(&dP,  sizeof(float)*nScores)!=hipSuccess){ hipFree(dS); return Status::RuntimeError; }
  if (hipMalloc(&dKt, sizeof(float)*(size_t)D*(size_t)N)!=hipSuccess){ hipFree(dS); hipFree(dP); return Status::RuntimeError; }

  GemmAttrs g{}; g.act=ActKind::None; g.with_bias=false;
  const float scale = (attrs.scale!=0.f) ? attrs.scale : (1.f/std::sqrt((float)D));
  hipStream_t s = to_cuda(stream);

  auto slice2d = [](const Tensor& T, int b, int h, int R, int C)->Tensor{
    size_t offset = ((size_t)b*T.desc.shape[1] + h) * (size_t)R*C;
    TensorDesc d{}; d.dtype=DType::F32; d.layout=Layout::RowMajor; d.shape={R,C}; d.stride={C,1};
    return Tensor{ (void*)((float*)T.data + offset), d, Device::CUDA, T.device_index };
  };

  Tensor S4{ dS, {DType::F32, Layout::RowMajor, {B,H,M,N}, {H*M*N, M*N, N, 1}}, Device::CUDA, Q.device_index };
  Tensor P4{ dP, {DType::F32, Layout::RowMajor, {B,H,M,N}, {H*M*N, M*N, N, 1}}, Device::CUDA, Q.device_index };

  // ---- Step 1: S = Q @ K^T ----
  dim3 blk(32, 8), grdKT((N + blk.x - 1)/blk.x, (D + blk.y - 1)/blk.y);

  for (int b=0;b<B;++b){
    for (int h=0; h<H; ++h){
      Tensor Q2 = slice2d(Q, b,h,M,D);
      Tensor K2 = slice2d(K, b,h,N,D);

      // K(N,D) -> Kt(D,N)
      const float* Kptr = static_cast<const float*>(K2.data);
      transpose_rm_f32<<<grdKT, blk, 0, s>>>(Kptr, dKt, /*R*/N, /*C*/D);
      if (hipPeekAtLastError()!=hipSuccess){ hipFree(dKt); hipFree(dS); hipFree(dP); return Status::RuntimeError; }

      TensorDesc kt_d{}; kt_d.dtype=DType::F32; kt_d.layout=Layout::RowMajor; kt_d.shape={D,N}; kt_d.stride={N,1};
      Tensor Kt{ dKt, kt_d, Device::CUDA, Q.device_index };

      // S = Q @ Kt
      Tensor S2 = slice2d(S4, b,h,M,N);
      if (ops::gemm_run(Q2, Kt, nullptr, S2, g, stream)!=0){ hipFree(dKt); hipFree(dS); hipFree(dP); return Status::RuntimeError; }

      // scale: S *= scale
      long long n = (long long)M*(long long)N; int BS=256, GRID=(int)((n+BS-1)/BS);
      scale_kernel<<<GRID,BS,0,s>>>((float*)S2.data, n, scale);
      if (hipPeekAtLastError()!=hipSuccess){ hipFree(dKt); hipFree(dS); hipFree(dP); return Status::RuntimeError; }
    }
  }

  // ---- Step 2: causal mask ----
  if (attrs.causal){
    int BS = 256;
    size_t total = (size_t)B * H * M * N;
    int GRID = (int)((total + BS - 1) / BS);
    causal_mask_add_kernel<<<GRID, BS, 0, s>>>(dS, B, H, M, N, -1e9f);
    if (hipPeekAtLastError()!=hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP);
      return Status::RuntimeError;
    }
  }

  // ---- Step 2.5: external mask ----
  if (mask) {
    if (mask->desc.shape.size()!=4 ||
        (int)mask->desc.shape[0]!=B || (int)mask->desc.shape[1]!=1 ||
        (int)mask->desc.shape[2]!=M || (int)mask->desc.shape[3]!=N) {
      hipFree(dKt); hipFree(dS); hipFree(dP);
      return Status::ShapeMismatch;
    }

    int BS = 256;
    size_t total = (size_t)B * H * M * N;
    int GRID = (int)((total + BS - 1) / BS);

    switch (mask->desc.dtype) {
      case DType::I8:
        add_mask_i8_kernel<<<GRID, BS, 0, s>>>(dS, (const int8_t*)mask->data, B, H, M, N, -1e9f);
        break;
      case DType::I32:
        add_mask_i32_kernel<<<GRID, BS, 0, s>>>(dS, (const int32_t*)mask->data, B, H, M, N, -1e9f);
        break;
      case DType::F32:
        add_mask_f32_kernel<<<GRID, BS, 0, s>>>(dS, (const float*)mask->data, B, H, M, N);
        break;
      default:
        hipFree(dKt); hipFree(dS); hipFree(dP);
        return Status::Invalid;
    }
    if (hipPeekAtLastError()!=hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP);
      return Status::RuntimeError;
    }
  }

  // ---- Step 3/4: P = softmax(S) [+ dropout] ----
  {
    SoftmaxAttrs sa{}; sa.scale=1.f; sa.log=false;
    for (int b=0; b<B; ++b){
      for (int h=0; h<H; ++h){
        Tensor S2{ (char*)dS + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                   {DType::F32, Layout::RowMajor, {M,N}, {N,1}}, Device::CUDA, Q.device_index };
        Tensor P2{ (char*)dP + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                   {DType::F32, Layout::RowMajor, {M,N}, {N,1}}, Device::CUDA, Q.device_index };
        if (ops::softmax_run(S2, /*mask*/nullptr, P2, sa, stream)!=0){ hipFree(dKt); hipFree(dS); hipFree(dP); return Status::RuntimeError; }

        if (attrs.dropout_p > 0.f){
          DropoutAttrs da{}; da.p=attrs.dropout_p; da.scale_in_train=attrs.scale_in_train; da.seed=attrs.seed;
          if (ops::dropout_run(P2, P2, /*mask*/nullptr, da, stream)!=0){ hipFree(dKt); hipFree(dS); hipFree(dP); return Status::RuntimeError; }
        }

        // ---- Step 5: Y = P @ V ----
        Tensor V2 = slice2d(V, b,h,N,D);
        Tensor Y2 = slice2d(Y, b,h,M,D);
        if (ops::gemm_run(P2, V2, nullptr, Y2, g, stream)!=0){ hipFree(dKt); hipFree(dS); hipFree(dP); return Status::RuntimeError; }
      }
    }
  }

  hipFree(dKt); hipFree(dS); hipFree(dP);
  return (hipPeekAtLastError()==hipSuccess) ? Status::Ok : Status::RuntimeError;
}

// 런타임 에러 리턴 헬퍼: 태그와 CUDA 에러 문자열(있다면) 출력
static inline ai::Status RTERR(const char* tag, hipError_t err = hipSuccess) {
  if (err != hipSuccess) {
    fprintf(stderr, "[SDPA-BWD][RuntimeError] at %s | cuda: %s\n", tag, hipGetErrorString(err));
  } else {
    fprintf(stderr, "[SDPA-BWD][RuntimeError] at %s\n", tag);
  }
  return ai::Status::RuntimeError;
}

ai::Status SDPACudaBackwardLaunch(const ai::Tensor& Q, const ai::Tensor& K, const ai::Tensor& V,
                                  const ai::Tensor& dY,
                                  const ai::Tensor* mask, 
                                  ai::Tensor* dQ, ai::Tensor* dK, ai::Tensor* dV,
                                  const ai::SDPAAttrs& a, ai::StreamHandle stream)
{
  // 최소 정책: 셋 다 null이면 안 됨
  if (!dQ && !dK && !dV) { fprintf(stderr, "[SDPA-BWD] invalid: all outputs null\n"); return ai::Status::Invalid; }

  // 타입/차원 체크
  if (!is4d_f32_cuda(Q) || !is4d_f32_cuda(K) || !is4d_f32_cuda(V) || !is4d_f32_cuda(dY)) {
    fprintf(stderr, "[SDPA-BWD] invalid: dtype/layout/device/ndim check failed for inputs\n");
    return ai::Status::Invalid;
  }
  if (dQ && !is4d_f32_cuda(*dQ)) { fprintf(stderr, "[SDPA-BWD] invalid: dQ bad\n"); return ai::Status::Invalid; }
  if (dK && !is4d_f32_cuda(*dK)) { fprintf(stderr, "[SDPA-BWD] invalid: dK bad\n"); return ai::Status::Invalid; }
  if (dV && !is4d_f32_cuda(*dV)) { fprintf(stderr, "[SDPA-BWD] invalid: dV bad\n"); return ai::Status::Invalid; }

  const int B = (int)Q.desc.shape[0];
  const int H = (int)Q.desc.shape[1];
  const int M = (int)Q.desc.shape[2];
  const int D = (int)Q.desc.shape[3];
  const int N = (int)K.desc.shape[2];

  if ((int)K.desc.shape[0]!=B || (int)K.desc.shape[1]!=H || (int)K.desc.shape[3]!=D) {
    fprintf(stderr, "[SDPA-BWD] shape mismatch: K\n"); return ai::Status::ShapeMismatch;
  }
  if ((int)V.desc.shape[0]!=B || (int)V.desc.shape[1]!=H || (int)V.desc.shape[2]!=N || (int)V.desc.shape[3]!=D) {
    fprintf(stderr, "[SDPA-BWD] shape mismatch: V\n"); return ai::Status::ShapeMismatch;
  }
  if ((int)dY.desc.shape[0]!=B || (int)dY.desc.shape[1]!=H || (int)dY.desc.shape[2]!=M || (int)dY.desc.shape[3]!=D) {
    fprintf(stderr, "[SDPA-BWD] shape mismatch: dY\n"); return ai::Status::ShapeMismatch;
  }
  if (dQ && dQ->desc.shape!=Q.desc.shape) { fprintf(stderr, "[SDPA-BWD] shape mismatch: dQ vs Q\n"); return ai::Status::ShapeMismatch; }
  if (dK && dK->desc.shape!=K.desc.shape) { fprintf(stderr, "[SDPA-BWD] shape mismatch: dK vs K\n"); return ai::Status::ShapeMismatch; }
  if (dV && dV->desc.shape!=V.desc.shape) { fprintf(stderr, "[SDPA-BWD] shape mismatch: dV vs V\n"); return ai::Status::ShapeMismatch; }

  if (a.dropout_p != 0.f) { fprintf(stderr, "[SDPA-BWD] invalid: dropout not supported yet\n"); return ai::Status::Invalid; }

  hipStream_t s = to_cuda(stream);

  // workspace: S,P,gP,gS (all [B,H,M,N])
  size_t nScores = (size_t)B*H*M*N;
  float *dS=nullptr, *dP=nullptr, *dgP=nullptr, *dgS=nullptr;
  if (hipMalloc(&dS,  sizeof(float)*nScores)!=hipSuccess) return RTERR("hipMalloc dS");
  if (hipMalloc(&dP,  sizeof(float)*nScores)!=hipSuccess){ hipFree(dS); return RTERR("hipMalloc dP"); }
  if (hipMalloc(&dgP, sizeof(float)*nScores)!=hipSuccess){ hipFree(dP); hipFree(dS); return RTERR("hipMalloc dgP"); }
  if (hipMalloc(&dgS, sizeof(float)*nScores)!=hipSuccess){ hipFree(dgP); hipFree(dP); hipFree(dS); return RTERR("hipMalloc dgS"); }

  ai::GemmAttrs g{}; g.act=ai::ActKind::None; g.with_bias=false;
  float scale = a.scale; if (scale==0.f) scale = 1.f / std::sqrt((float)D);

  // 공용 전치 버퍼 (Kᵀ / Vᵀ 용)
  float* dKt = nullptr;
  if (hipMalloc(&dKt, sizeof(float) * (size_t)D * (size_t)N) != hipSuccess) {
    hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
    return RTERR("hipMalloc K^T (step1)");
  }
  
  // 1) S = scale * (Q @ K^T)
  for (int b=0; b<B; ++b){
    for (int h=0; h<H; ++h){
      ai::Tensor tQ{ (char*)Q.data + ((size_t)((((size_t)b*H+h)*M)*D))*sizeof(float),
                     {ai::DType::F32, ai::Layout::RowMajor, {M,D}, {D,1}}, ai::Device::CUDA, Q.device_index };
      ai::Tensor tK{ (char*)K.data + ((size_t)((((size_t)b*H+h)*N)*D))*sizeof(float),
                     {ai::DType::F32, ai::Layout::RowMajor, {N,D}, {D,1}}, ai::Device::CUDA, K.device_index };
      ai::Tensor tS{ (char*)dS  + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                     {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };

      // (N×D) -> (D×N)
      {
        dim3 blk(32, 8);
        dim3 grd((D + blk.x - 1) / blk.x, (N + blk.y - 1) / blk.y);
        transpose_rm_f32<<<grd, blk, 0, s>>>((const float*)tK.data, dKt, /*R*/N, /*C*/D);
        if (hipPeekAtLastError()!=hipSuccess){
          hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("transpose K->Kt (step1)", hipGetLastError());
        }
      }
      ai::Tensor tKt{ dKt,
                      {ai::DType::F32, ai::Layout::RowMajor, {D,N}, {N,1}},
                      ai::Device::CUDA, K.device_index };

      if (ai::ops::gemm_run(tQ, tKt, nullptr, tS, g, stream)!=0){
        hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
        return RTERR("gemm S = Q @ K^T");
      }

      long long n = (long long)M*(long long)N; int BS=256, GRID=(int)((n+BS-1)/BS);
      scale_kernel<<<GRID,BS,0,s>>>((float*)tS.data, n, scale);
      if (hipPeekAtLastError()!=hipSuccess){
        hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
        return RTERR("scale S (step1)", hipGetLastError());
      }
    }
  }

  // 2) causal mask
  if (a.causal){
    int BS=256, GRID=(int)((nScores + BS - 1)/BS);
    causal_mask_add_kernel<<<GRID,BS,0,s>>>(dS, B,H,M,N, -1e9f);
    if (hipPeekAtLastError()!=hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
      return RTERR("causal_mask_add_kernel", hipGetLastError());
    }
  }

  // 3) P = softmax(S)
  {
    ai::SoftmaxAttrs sa{}; sa.scale=1.f; sa.log=false;
    for (int b=0; b<B; ++b){
      for (int h=0; h<H; ++h){
        ai::Tensor tS{ (char*)dS + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                       {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tP{ (char*)dP + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                       {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        if (ai::ops::softmax_run(tS, /*mask*/nullptr, tP, sa, stream)!=0){
          hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("softmax forward P");
        }
      }
    }
  }

  // 4) dV = P^T @ dY
  if (dV){
    float* dYt_buf = nullptr;
    if (hipMalloc(&dYt_buf, sizeof(float) * (size_t)D * (size_t)M) != hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
      return RTERR("hipMalloc dYt_buf (dV path)");
    }

    for (int b=0; b<B; ++b){
      for (int h=0; h<H; ++h){
        ai::Tensor tP{  (char*)dP + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tdY{ (char*)dY.data + ((size_t)((((size_t)b*H+h)*M)*D))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,D}, {D,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tdV{ (char*)dV->data + ((size_t)((((size_t)b*H+h)*N)*D))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {N,D}, {D,1}}, ai::Device::CUDA, dV->device_index };

        // (M×D) -> (D×M)
        {
          dim3 blk(32,8);
          dim3 grd((D + blk.x - 1)/blk.x, (M + blk.y - 1)/blk.y);
          transpose_rm_f32<<<grd, blk, 0, s>>>((const float*)tdY.data, dYt_buf, /*R*/M, /*C*/D);
          if (hipPeekAtLastError()!=hipSuccess){
            hipFree(dYt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
            return RTERR("transpose dY->dYt (dV path)", hipGetLastError());
          }
        }
        ai::Tensor tYt{ dYt_buf, {ai::DType::F32, ai::Layout::RowMajor, {D,M}, {M,1}}, ai::Device::CUDA, Q.device_index };

        // T1 = dY^T(D×M) @ P(M×N) = (D×N)
        float* dT1=nullptr;
        if (hipMalloc(&dT1, sizeof(float)*(size_t)D*(size_t)N)!=hipSuccess){
          hipFree(dYt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("hipMalloc T1 (dV path)");
        }
        ai::Tensor tT1{ dT1, {ai::DType::F32, ai::Layout::RowMajor, {D,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        if (ai::ops::gemm_run(tYt, tP, nullptr, tT1, g, stream)!=0){
          hipFree(dT1); hipFree(dYt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("gemm T1 = dY^T @ P");
        }

        // tdV = T1^T
        dim3 blk2(32,8), grd2((N + blk2.x -1)/blk2.x, (D + blk2.y -1)/blk2.y);
        transpose_rm_f32<<<grd2,blk2,0,s>>>(dT1, (float*)tdV.data, /*R*/D, /*C*/N);
        hipFree(dT1);
        if (hipPeekAtLastError()!=hipSuccess){
          hipFree(dYt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("transpose tdV", hipGetLastError());
        }
      }
    }
    hipFree(dYt_buf);
  }

  // 5) gP = dY @ V^T
  for (int b=0; b<B; ++b){
    for (int h=0; h<H; ++h){
      ai::Tensor tdY{ (char*)dY.data + ((size_t)((((size_t)b*H+h)*M)*D))*sizeof(float),
                      {ai::DType::F32, ai::Layout::RowMajor, {M,D}, {D,1}}, ai::Device::CUDA, Q.device_index };
      ai::Tensor tV { (char*)V.data  + ((size_t)((((size_t)b*H+h)*N)*D))*sizeof(float),
                      {ai::DType::F32, ai::Layout::RowMajor, {N,D}, {D,1}}, ai::Device::CUDA, V.device_index };
      ai::Tensor tgP{ (char*)dgP     + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                      {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };

      // V(N×D) -> Vt(D×N) in dKt (재사용)
      {
        dim3 blk(32, 8);
        dim3 grd((D + blk.x - 1) / blk.x, (N + blk.y - 1) / blk.y);
        transpose_rm_f32<<<grd, blk, 0, s>>>((const float*)tV.data, dKt, /*R*/N, /*C*/D);
        if (hipPeekAtLastError()!=hipSuccess){
          hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("transpose V->Vt (step5)", hipGetLastError());
        }
      }
      ai::Tensor tVt{ dKt,
                      {ai::DType::F32, ai::Layout::RowMajor, {D,N}, {N,1}},
                      ai::Device::CUDA, V.device_index };

      if (ai::ops::gemm_run(tdY, tVt, nullptr, tgP, g, stream)!=0){
        hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
        return RTERR("gemm gP = dY @ V^T");
      }
    }
  }

  // 6) gS = softmax_backward(P, gP)
  {
    ai::SoftmaxAttrs sa{}; sa.scale=1.f; sa.log=false;
    for (int b=0; b<B; ++b){
      for (int h=0; h<H; ++h){
        ai::Tensor tP { (char*)dP  + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tgP{ (char*)dgP + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tgS{ (char*)dgS + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        if (ai::ops::softmax_backward_run(tP, tgP, tgS, sa, stream)!=0){
          hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("softmax backward gS");
        }
      }
    }
  }

  // 7) causal ⇒ gS 상삼각 0
  if (a.causal){
    int BS=256, GRID=(int)((nScores + BS - 1)/BS);
    causal_gs_zero_kernel<<<GRID,BS,0,s>>>(dgS, B,H,M,N);
    if (hipPeekAtLastError()!=hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
      return RTERR("causal_gs_zero_kernel", hipGetLastError());
    }
  }

    // 7.5) external mask ⇒ gS = 0 on masked positions
  if (mask) {
    if (mask->desc.shape.size()!=4 ||
        (int)mask->desc.shape[0]!=B || (int)mask->desc.shape[1]!=1 ||
        (int)mask->desc.shape[2]!=M || (int)mask->desc.shape[3]!=N) {
      hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
      return RTERR("mask shape mismatch (bwd)");
    }
    int BS=256, GRID=(int)(((size_t)B*H*M*N + BS - 1)/BS);
    switch (mask->desc.dtype) {
      case DType::I8:
        zero_gs_mask_i8_kernel<<<GRID,BS,0,s>>>(dgS, (const int8_t*)mask->data, B,H,M,N);
        break;
      case DType::I32:
        zero_gs_mask_i32_kernel<<<GRID,BS,0,s>>>(dgS, (const int32_t*)mask->data, B,H,M,N);
        break;
      case DType::F32:
        zero_gs_mask_f32_kernel<<<GRID,BS,0,s>>>(dgS, (const float*)mask->data, B,H,M,N);
        break;
      default:
        hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
        return RTERR("mask dtype invalid (bwd)");
    }
    if (hipPeekAtLastError()!=hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
      return RTERR("zero_gs_mask_*_kernel", hipGetLastError());
    }
  }

  // 8) dQ = scale * (gS @ K)
  if (dQ){
    for (int b=0; b<B; ++b){
      for (int h=0; h<H; ++h){
        ai::Tensor tgS{ (char*)dgS + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tK { (char*)K.data + ((size_t)((((size_t)b*H+h)*N)*D))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {N,D}, {D,1}}, ai::Device::CUDA, K.device_index };
        ai::Tensor tdQ{ (char*)dQ->data + ((size_t)((((size_t)b*H+h)*M)*D))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,D}, {D,1}}, ai::Device::CUDA, dQ->device_index };
        if (ai::ops::gemm_run(tgS, tK, nullptr, tdQ, g, stream)!=0){
          hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("gemm dQ = gS @ K");
        }
        long long n = (long long)M*(long long)D; int BS=256, GRID=(int)((n+BS-1)/BS);
        scale_kernel<<<GRID,BS,0,s>>>((float*)tdQ.data, n, scale);
        if (hipPeekAtLastError()!=hipSuccess){
          hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("scale dQ", hipGetLastError());
        }
      }
    }
  }

  // 9) dK = scale * (Q^T @ gS)^T
  if (dK){
    float* Qt_buf = nullptr;
    if (hipMalloc(&Qt_buf, sizeof(float) * (size_t)D * (size_t)M) != hipSuccess){
      hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
      return RTERR("hipMalloc Qt_buf (dK path)");
    }

    for (int b=0; b<B; ++b){
      for (int h=0; h<H; ++h){
        ai::Tensor tQ { (char*)Q.data + ((size_t)((((size_t)b*H+h)*M)*D))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,D}, {D,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tgS{ (char*)dgS    + ((size_t)((((size_t)b*H+h)*M)*N))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {M,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        ai::Tensor tdK{ (char*)dK->data + ((size_t)((((size_t)b*H+h)*N)*D))*sizeof(float),
                        {ai::DType::F32, ai::Layout::RowMajor, {N,D}, {D,1}}, ai::Device::CUDA, dK->device_index };

        // Q(M×D) -> Q^T(D×M)
        {
          dim3 blk(32,8);
          dim3 grd((D + blk.x - 1)/blk.x, (M + blk.y - 1)/blk.y);
          transpose_rm_f32<<<grd, blk, 0, s>>>((const float*)tQ.data, Qt_buf, /*R*/M, /*C*/D);
          if (hipPeekAtLastError()!=hipSuccess){
            hipFree(Qt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
            return RTERR("transpose Q->Qt (dK path)", hipGetLastError());
          }
        }
        ai::Tensor tQt{ Qt_buf, {ai::DType::F32, ai::Layout::RowMajor, {D,M}, {M,1}}, ai::Device::CUDA, Q.device_index };

        // T2 = Q^T(D×M) @ gS(M×N) = (D×N)
        float* dT2 = nullptr;
        if (hipMalloc(&dT2, sizeof(float)*(size_t)D*(size_t)N)!=hipSuccess){
          hipFree(Qt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("hipMalloc T2 (dK path)");
        }
        ai::Tensor tT2{ dT2, {ai::DType::F32, ai::Layout::RowMajor, {D,N}, {N,1}}, ai::Device::CUDA, Q.device_index };
        if (ai::ops::gemm_run(tQt, tgS, nullptr, tT2, g, stream)!=0){
          hipFree(dT2); hipFree(Qt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("gemm T2 = Q^T @ gS");
        }

        // dK = (T2)^T scaled
        dim3 blk2(32,8), grd2((N + blk2.x -1)/blk2.x, (D + blk2.y -1)/blk2.y);
        transpose_rm_f32<<<grd2,blk2,0,s>>>(dT2, (float*)tdK.data, /*R*/D, /*C*/N);
        hipFree(dT2);
        if (hipPeekAtLastError()!=hipSuccess){
          hipFree(Qt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("transpose tdK", hipGetLastError());
        }

        long long n = (long long)N*(long long)D;
        int BS=256, GRID=(int)((n+BS-1)/BS);
        scale_kernel<<<GRID,BS,0,s>>>((float*)tdK.data, n, scale);
        if (hipPeekAtLastError()!=hipSuccess){
          hipFree(Qt_buf); hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
          return RTERR("scale dK", hipGetLastError());
        }
      }
    }
    hipFree(Qt_buf);
  }

  // 최종 커널 에러 체크
  hipError_t e = hipPeekAtLastError();
  if (e != hipSuccess) {
    hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
    return RTERR("final hipPeekAtLastError", e);
  }

  hipFree(dKt); hipFree(dS); hipFree(dP); hipFree(dgP); hipFree(dgS);
  return ai::Status::Ok;
}

} // namespace ai
