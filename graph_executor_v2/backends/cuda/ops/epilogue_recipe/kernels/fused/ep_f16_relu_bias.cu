#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../epilogue_params.cuh"
#include "../functors/act_relu.cuh"
#include "../functors/dropout_philox.cuh"

extern "C" __global__
void ep_f16_relu_bias(EpParamsF16 P){
  PhiloxState st{P.seed, P.offset};
  int t = blockIdx.x*blockDim.x + threadIdx.x, T=P.M*P.N;
  for(int i=t; i<T; i += gridDim.x*blockDim.x){
    int m=i/P.N, n=i%P.N;
    int ix=m*P.ld_x+n, iy=m*P.ld_y+n;
    half v = relu_h(__hadd(P.x[ix], P.bias[n]));
    if (P.use_dropout) v = apply_dropout<half>(v, st, (unsigned long long)i, P.p_drop, P.keep_scale);
    float out = P.alpha*__half2float(v) + (P.beta!=0.f ? P.beta*__half2float(P.y[iy]) : 0.f);
    P.y[iy] = __float2half(out);
  }
}
