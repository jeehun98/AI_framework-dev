#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "backends/cuda/ops/memory/api.hpp"

namespace ai {

static inline hipStream_t to_cuda(StreamHandle h){ return (hipStream_t)h; }

template<int MAX_D=8>
__global__ void contiguous_copy_kernel(
  const float* __restrict__ x, float* __restrict__ y,
  int D,
  const int64_t* __restrict__ shape,
  const int64_t* __restrict__ stride_in,
  int64_t total)
{
  int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= total) return;

  // tid -> 좌표
  int64_t idx[MAX_D];
  int64_t t = tid;
  #pragma unroll
  for (int d=D-1; d>=0; --d){
    idx[d] = t % shape[d];
    t /= shape[d];
  }

  // 좌표 -> 입력 오프셋
  int64_t off = 0;
  #pragma unroll
  for (int d=0; d<D; ++d){
    off += idx[d] * stride_in[d];
  }

  y[tid] = x[off];
}

Status ContiguousCopyCudaLaunch(const Tensor& X, Tensor& Y, StreamHandle stream)
{
  // 제약: F32, RowMajor, 1<=D<=8
  if (X.desc.dtype != DType::F32 || Y.desc.dtype != DType::F32) return Status::Invalid;
  if (X.device != Device::CUDA || Y.device != Device::CUDA)     return Status::Invalid;
  if (X.desc.shape.size() != Y.desc.shape.size())               return Status::ShapeMismatch;

  const int D = (int)X.desc.shape.size();
  if (D < 1 || D > 8) return Status::Invalid;

  // shape 동일성만 보장(뷰/stride는 다를 수 있음)
  for (int i=0;i<D;++i){
    if (X.desc.shape[i] != Y.desc.shape[i]) return Status::ShapeMismatch;
  }

  // 메타 업로드
  int64_t h_shape[8], h_stride_in[8];
  int64_t total = 1;
  for (int i=0;i<D;++i){
    h_shape[i]     = X.desc.shape[i];
    h_stride_in[i] = X.desc.stride[i];
    total *= X.desc.shape[i];
  }

  int64_t *d_shape=nullptr, *d_stride_in=nullptr;
  hipError_t e;
  e = hipMalloc(&d_shape,     sizeof(int64_t)*D); if (e!=hipSuccess) return Status::RuntimeError;
  e = hipMalloc(&d_stride_in, sizeof(int64_t)*D); if (e!=hipSuccess){ hipFree(d_shape); return Status::RuntimeError; }
  e = hipMemcpyAsync(d_shape,     h_shape,     sizeof(int64_t)*D, hipMemcpyHostToDevice, to_cuda(stream)); if (e!=hipSuccess){ hipFree(d_shape); hipFree(d_stride_in); return Status::RuntimeError; }
  e = hipMemcpyAsync(d_stride_in, h_stride_in, sizeof(int64_t)*D, hipMemcpyHostToDevice, to_cuda(stream)); if (e!=hipSuccess){ hipFree(d_shape); hipFree(d_stride_in); return Status::RuntimeError; }

  // 런치
  const int BS = 256;
  dim3 block(BS), grid((int)((total + BS - 1)/BS));
  contiguous_copy_kernel<8><<<grid, block, 0, to_cuda(stream)>>>(
    static_cast<const float*>(X.data),
    static_cast<float*>(Y.data),
    D, d_shape, d_stride_in, total
  );

  // 정리
  hipFree(d_shape);
  hipFree(d_stride_in);
  return (hipPeekAtLastError()==hipSuccess) ? Status::Ok : Status::RuntimeError;
}

} // namespace ai
